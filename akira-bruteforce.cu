#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <fstream>
#include <unistd.h>
//stat
#include <sys/types.h>
#include <sys/stat.h>
//mmap
#include <sys/mman.h>
#include <fcntl.h>
#include "json.hpp"
#include "test-ts.h"
#include "akira-bruteforce.h"
#include "chacha8.h"

//by design, max matches is 127
#define MAX_MATCHES 32

using json = nlohmann::json;

int gpuIndex = 0;

#define SHA256_DIGEST_SIZE 32

// --- SHA-256 Device Implementation ---

// Rotate right.
__device__ __forceinline__ uint32_t rotr(uint32_t x, uint32_t n)
{
    //return (x >> n) | (x << (32 - n));
        return __funnelshift_r(x, x, n );

}


#define SHR(x, n) ((x) >> (n))
#define Ch(x, y, z) (((x) & (y)) ^ ((~(x)) & (z)))
#define Maj(x, y, z) (((x) & (y)) ^ ((x) & (z)) ^ ((y) & (z)))
#define Sigma0(x) (rotr(x, 2) ^ rotr(x, 13) ^ rotr(x, 22))
#define Sigma1(x) (rotr(x, 6) ^ rotr(x, 11) ^ rotr(x, 25))
#define sigma0(x) (rotr(x, 7) ^ rotr(x, 18) ^ SHR(x, 3))
#define sigma1(x) (rotr(x, 17) ^ rotr(x, 19) ^ SHR(x, 10))

// Process one 64-byte block.
__device__ __forceinline__ void sha256_transform(const uint8_t *data, uint32_t state[8], const uint32_t *k)
{
    uint32_t w[64];
#pragma unroll
    for (int i = 0; i < 16; i++)
    {
        w[i] = ((uint32_t)data[i * 4] << 24) |
               ((uint32_t)data[i * 4 + 1] << 16) |
               ((uint32_t)data[i * 4 + 2] << 8) |
               ((uint32_t)data[i * 4 + 3]);
    }
    for (int i = 16; i < 64; i++)
    {
        w[i] = sigma1(w[i - 2]) + w[i - 7] + sigma0(w[i - 15]) + w[i - 16];
    }

    uint32_t a = state[0];
    uint32_t b = state[1];
    uint32_t c = state[2];
    uint32_t d = state[3];
    uint32_t e = state[4];
    uint32_t f = state[5];
    uint32_t g = state[6];
    uint32_t h = state[7];

    for (int i = 0; i < 64; i++)
    {
        uint32_t T1 = h + Sigma1(e) + Ch(e, f, g) + k[i] + w[i];
        uint32_t T2 = Sigma0(a) + Maj(a, b, c);
        h = g;
        g = f;
        f = e;
        e = d + T1;
        d = c;
        c = b;
        b = a;
        a = T1 + T2;
    }

    state[0] += a;
    state[1] += b;
    state[2] += c;
    state[3] += d;
    state[4] += e;
    state[5] += f;
    state[6] += g;
    state[7] += h;
}

// For a fixed 19-byte input, the padded message is exactly 64 bytes.
// Padding: message (19 bytes) || 0x80 || (zeros up to byte 56) || [0x0000000000000098]
// (19*8 = 152, or 0x98)
__device__ void sha256_hash_19(const uint8_t *msg, uint8_t *digest, const uint32_t *k)
{
    uint32_t state[8] = {
        0x6a09e667,
        0xbb67ae85,
        0x3c6ef372,
        0xa54ff53a,
        0x510e527f,
        0x9b05688c,
        0x1f83d9ab,
        0x5be0cd19};

    
    uint8_t block[64];

#pragma unroll
    for (int i = 0; i < 19; i++)
        block[i] = msg[i];
    block[19] = 0x80;
#pragma unroll
    for (int i = 20; i < 56; i++)
        block[i] = 0;
    // Append message length in bits: 152 = 0x0000000000000098 (big-endian)
    block[56] = 0;
    block[57] = 0;
    block[58] = 0;
    block[59] = 0;
    block[60] = 0;
    block[61] = 0;
    block[62] = 0;
    block[63] = 152;

    sha256_transform(block, state, k);

#pragma unroll
    for (int i = 0; i < 8; i++)
    {
        digest[i * 4] = (state[i] >> 24) & 0xff;
        digest[i * 4 + 1] = (state[i] >> 16) & 0xff;
        digest[i * 4 + 2] = (state[i] >> 8) & 0xff;
        digest[i * 4 + 3] = state[i] & 0xff;
    }
}

// For a fixed 68-byte input, the padded message consists of 2 blocks.
// Block 1: first 64 bytes of the input.
// Block 2: remaining 4 bytes || 0x80 || zeros until byte 56 || [bit length = 544 bits]
__device__ void sha256_hash_68(const uint8_t * msg, uint8_t *digest, const uint32_t *  k)
{
    uint32_t state[8] = {
        0x6a09e667,
        0xbb67ae85,
        0x3c6ef372,
        0xa54ff53a,
        0x510e527f,
        0x9b05688c,
        0x1f83d9ab,
        0x5be0cd19};

    uint8_t block2[64];

    // Block 1: copy first 64 bytes.

    //uint8_t block[64];
#pragma unroll	    
    for (int i = 0; i < 64; i++)
	    block2[i] = msg[i];
    sha256_transform(block2, state, k);
    

    // Block 2:
    // First 4 bytes: remainder of message.
#pragma unroll
    for (int i = 0; i < 4; i++)
        block2[i] = msg[64 + i];
    block2[4] = 0x80;
#pragma unroll
    for (int i = 5; i < 56; i++)
        block2[i] = 0;
    // Append message length in bits: 68*8 = 544 = 0x0000000000000220 (big-endian)
    block2[56] = 0;
    block2[57] = 0;
    block2[58] = 0;
    block2[59] = 0;
    block2[60] = 0;
    block2[61] = 0;
    block2[62] = 0x02;
    block2[63] = 0x20;

    sha256_transform(block2, state, k);

#pragma unroll
    for (int i = 0; i < 8; i++)
    {
        digest[i * 4] = (state[i] >> 24) & 0xff;
        digest[i * 4 + 1] = (state[i] >> 16) & 0xff;
        digest[i * 4 + 2] = (state[i] >> 8) & 0xff;
        digest[i * 4 + 3] = state[i] & 0xff;
    }
}

// Minimal AES-256 context and helper functions.
typedef struct
{
    uint32_t rk[60]; // 60 words for 14 rounds + initial key
} aes256_ctx;

__device__ __forceinline__ uint32_t SubWord(uint32_t word, const uint8_t *sbox)
{
    return ((uint32_t)sbox[(word >> 24)] << 24) |
           ((uint32_t)sbox[(word >> 16) & 0xff] << 16) |
           ((uint32_t)sbox[(word >> 8) & 0xff] << 8) |
           ((uint32_t)sbox[word & 0xff]);
}

__device__ inline uint32_t RotWord(uint32_t word)
{
    //return (word << 8) | (word >> 24);
    return __funnelshift_l(word, word, 8);

}

// Key expansion for AES-256.
__device__ void aes256_set_encrypt_key(const uint8_t userKey[32], aes256_ctx *ctx, const uint8_t *sbox, const uint32_t *Rcon)
{
    int i = 0;
    // Copy the 256-bit key into the first 8 words.
    for (i = 0; i < 8; i++)
    {
        ctx->rk[i] = ((uint32_t)userKey[4 * i] << 24) |
                     ((uint32_t)userKey[4 * i + 1] << 16) |
                     ((uint32_t)userKey[4 * i + 2] << 8) |
                     ((uint32_t)userKey[4 * i + 3]);
    }
    int rcon_i = 0;
  
    for (i = 8; i < 60; i++)
    {
	uint32_t temp = ctx->rk[i - 1];
        if ((i % 8) == 0)
        {
            temp = SubWord(RotWord(temp), sbox) ^ Rcon[rcon_i++];
        }
        else if ((i % 8) == 4)
        {
            temp = SubWord(temp, sbox);
        }
        ctx->rk[i] = ctx->rk[i - 8] ^ temp;
    }
}


// AES block encryption for a single 16-byte block.
__device__ void aes256_encrypt(const aes256_ctx *ctx, const uint8_t in[16], uint8_t out[16], const uint8_t *sbox, const uint8_t *xtime)
{
    uint8_t state[16];
    // Copy input into state.
    for (int i = 0; i < 16; i++)
    {
        state[i] = in[i];
    }
    // Initial AddRoundKey.
    for (int i = 0; i < 4; i++)
    {
        uint32_t rk = ctx->rk[i];
        state[4 * i + 0] ^= (uint8_t)(rk >> 24);
        state[4 * i + 1] ^= (uint8_t)(rk >> 16);
        state[4 * i + 2] ^= (uint8_t)(rk >> 8);
        state[4 * i + 3] ^= (uint8_t)(rk);
    }
    // Main rounds.
    for (int round = 1; round < 14; round++)
    {
        // SubBytes.
        for (int i = 0; i < 16; i++)
            state[i] = sbox[state[i]];
        // ShiftRows.
        uint8_t tmp[16];
        tmp[0] = state[0];
        tmp[1] = state[5];
        tmp[2] = state[10];
        tmp[3] = state[15];

        tmp[4] = state[4];
        tmp[5] = state[9];
        tmp[6] = state[14];
        tmp[7] = state[3];

        tmp[8] = state[8];
        tmp[9] = state[13];
        tmp[10] = state[2];
        tmp[11] = state[7];

        tmp[12] = state[12];
        tmp[13] = state[1];
        tmp[14] = state[6];
        tmp[15] = state[11];

        // MixColumns.
        for (int i = 0; i < 4; i++)
        {
            int col = 4 * i;
            uint8_t a0 = tmp[col + 0], a1 = tmp[col + 1],
                    a2 = tmp[col + 2], a3 = tmp[col + 3];
            uint8_t r0 = xtime[a0] ^ (a1 ^ xtime[a1]) ^ a2 ^ a3;
            uint8_t r1 = a0 ^ xtime[a1] ^ (a2 ^ xtime[a2]) ^ a3;
            uint8_t r2 = a0 ^ a1 ^ xtime[a2] ^ (a3 ^ xtime[a3]);
            uint8_t r3 = (a0 ^ xtime[a0]) ^ a1 ^ a2 ^ xtime[a3];
            tmp[col + 0] = r0;
            tmp[col + 1] = r1;
            tmp[col + 2] = r2;
            tmp[col + 3] = r3;
        }
        // Copy back to state.
        for (int i = 0; i < 16; i++)
            state[i] = tmp[i];
        // AddRoundKey.
        for (int i = 0; i < 4; i++)
        {
            uint32_t rk = ctx->rk[round * 4 + i];
            state[4 * i + 0] ^= (uint8_t)(rk >> 24);
            state[4 * i + 1] ^= (uint8_t)(rk >> 16);
            state[4 * i + 2] ^= (uint8_t)(rk >> 8);
            state[4 * i + 3] ^= (uint8_t)(rk);
        }
    }
    // Final round (no MixColumns).
    // SubBytes.
    for (int i = 0; i < 16; i++)
        state[i] = sbox[state[i]];
    // ShiftRows.
    uint8_t tmp[16];
    tmp[0] = state[0];
    tmp[1] = state[5];
    tmp[2] = state[10];
    tmp[3] = state[15];

    tmp[4] = state[4];
    tmp[5] = state[9];
    tmp[6] = state[14];
    tmp[7] = state[3];

    tmp[8] = state[8];
    tmp[9] = state[13];
    tmp[10] = state[2];
    tmp[11] = state[7];

    tmp[12] = state[12];
    tmp[13] = state[1];
    tmp[14] = state[6];
    tmp[15] = state[11];
    // Final AddRoundKey.
    for (int i = 0; i < 4; i++)
    {
        uint32_t rk = ctx->rk[14 * 4 + i];
        tmp[4 * i + 0] ^= (uint8_t)(rk >> 24);
        tmp[4 * i + 1] ^= (uint8_t)(rk >> 16);
        tmp[4 * i + 2] ^= (uint8_t)(rk >> 8);
        tmp[4 * i + 3] ^= (uint8_t)(rk);
    }
    // Write result.
    for (int i = 0; i < 16; i++)
        out[i] = tmp[i];
}

// kcipher2 state structure.
typedef struct
{
    unsigned int A[5];
    unsigned int B[11];
    unsigned int L1, R1, L2, R2;
} kcipher2_state;

//---------------------------------------------------------------------------
// Device functions for kcipher2
//---------------------------------------------------------------------------

__device__ __forceinline__ unsigned int nlf(unsigned int a, unsigned int b, unsigned int c, unsigned int d)
{
    return (a + b) ^ c ^ d;
}

__device__ __forceinline__ unsigned char gf_multiply_by_2(unsigned char t)
{
    // return gf2_table[t];
    unsigned int lq = t << 1;
    if (lq & 0x100)
        lq ^= 0x011B;
    return ((unsigned char)lq) ^ 0xFF;
}

__device__ __forceinline__ unsigned char gf_multiply_by_3(unsigned char t)
{
    //    return gf3_table[t];
    unsigned int lq = (t << 1) ^ t;
    if (lq & 0x100)
        lq ^= 0x011B;
    return ((unsigned char)lq) ^ 0xFF;
}

__device__ unsigned int sub_k2(unsigned int in)
{
    unsigned char w0 = in & 0xFF;
    unsigned char w1 = (in >> 8) & 0xFF;
    unsigned char w2 = (in >> 16) & 0xFF;
    unsigned char w3 = (in >> 24) & 0xFF;

    unsigned char t0 = d_s_box[w0];
    unsigned char t1 = d_s_box[w1];
    unsigned char t2 = d_s_box[w2];
    unsigned char t3 = d_s_box[w3];

    unsigned char q0 = gf_multiply_by_2(t0) ^ gf_multiply_by_3(t1) ^ t2 ^ t3;
    unsigned char q1 = t0 ^ gf_multiply_by_2(t1) ^ gf_multiply_by_3(t2) ^ t3;
    unsigned char q2 = t0 ^ t1 ^ gf_multiply_by_2(t2) ^ gf_multiply_by_3(t3);
    unsigned char q3 = gf_multiply_by_3(t0) ^ t1 ^ t2 ^ gf_multiply_by_2(t3);

    return ((unsigned int)q3 << 24) | ((unsigned int)q2 << 16) | ((unsigned int)q1 << 8) | q0;
}

__device__ void setup_state_values(const unsigned int *key, const unsigned int *iv, kcipher2_state *state)
{
    unsigned int IK[12];
    IK[0] = key[0];
    IK[1] = key[1];
    IK[2] = key[2];
    IK[3] = key[3];

    IK[4] = IK[0] ^ sub_k2((IK[3] << 8) ^ (IK[3] >> 24)) ^ 0x01000000;
    IK[5] = IK[1] ^ IK[4];
    IK[6] = IK[2] ^ IK[5];
    IK[7] = IK[3] ^ IK[6];
    IK[8] = IK[4] ^ sub_k2((IK[7] << 8) ^ (IK[7] >> 24)) ^ 0x02000000;

    IK[9] = IK[5] ^ IK[8];
    IK[10] = IK[6] ^ IK[9];
    IK[11] = IK[7] ^ IK[10];

    state->A[0] = IK[4];
    state->A[1] = IK[3];
    state->A[2] = IK[2];
    state->A[3] = IK[1];
    state->A[4] = IK[0];

    state->B[0] = IK[10];
    state->B[1] = IK[11];
    state->B[2] = iv[0];
    state->B[3] = iv[1];
    state->B[4] = IK[8];
    state->B[5] = IK[9];
    state->B[6] = iv[2];
    state->B[7] = iv[3];
    state->B[8] = IK[7];
    state->B[9] = IK[5];
    state->B[10] = IK[6];

    state->L1 = state->R1 = state->L2 = state->R2 = 0x00000000;
}

__device__ void next_INIT(kcipher2_state *state)
{
    unsigned int temp2;
    unsigned int nL1 = sub_k2(state->R2 + state->B[4]);
    unsigned int nR1 = sub_k2(state->L2 + state->B[9]);
    unsigned int nL2 = sub_k2(state->L1);
    unsigned int nR2 = sub_k2(state->R1);

    unsigned int nA[5];
    nA[0] = state->A[1];
    nA[1] = state->A[2];
    nA[2] = state->A[3];
    nA[3] = state->A[4];

    unsigned int nB[11];
    nB[0] = state->B[1];
    nB[1] = state->B[2];
    nB[2] = state->B[3];
    nB[3] = state->B[4];
    nB[4] = state->B[5];
    nB[5] = state->B[6];
    nB[6] = state->B[7];
    nB[7] = state->B[8];
    nB[8] = state->B[9];
    nB[9] = state->B[10];

    unsigned int temp1 = (state->A[0] << 8) ^ d_amul0[(state->A[0] >> 24) & 0xFF];
    nA[4] = temp1 ^ state->A[3];
    nA[4] ^= nlf(state->B[0], state->R2, state->R1, state->A[4]);

    if (state->A[2] & 0x40000000)
        temp1 = (state->B[0] << 8) ^ d_amul1[(state->B[0] >> 24) & 0xFF];
    else
        temp1 = (state->B[0] << 8) ^ d_amul2[(state->B[0] >> 24) & 0xFF];

    // branchless version (not faster)
    //  unsigned int mask = -(unsigned int)(!!(state->A[2] & 0x40000000));
    //  unsigned int candidate1 = (state->B[0] << 8) ^ d_amul1[(state->B[0] >> 24) & 0xFF];
    //  unsigned int candidate2 = (state->B[0] << 8) ^ d_amul2[(state->B[0] >> 24) & 0xFF];
    //  temp1 = candidate2 ^ ((candidate1 ^ candidate2) & mask);

    if (state->A[2] & 0x80000000)
        temp2 = (state->B[8] << 8) ^ d_amul3[(state->B[8] >> 24) & 0xFF];
    else
        temp2 = state->B[8];

    // branchless version (not faster)
    //     unsigned int mask2 = -(unsigned int)(!!(state->A[2] & 0x80000000));
    // unsigned int candidate3 = (state->B[8] << 8) ^ d_amul3[(state->B[8] >> 24) & 0xFF];
    // unsigned int candidate4 = state->B[8];
    // temp2 = candidate4 ^ ((candidate3 ^ candidate4) & mask2);

    nB[10] = temp1 ^ state->B[1] ^ state->B[6] ^ temp2;
    nB[10] ^= nlf(state->B[10], state->L2, state->L1, state->A[0]);

    state->A[0] = nA[0];
    state->A[1] = nA[1];
    state->A[2] = nA[2];
    state->A[3] = nA[3];
    state->A[4] = nA[4];

    state->B[0] = nB[0];
    state->B[1] = nB[1];
    state->B[2] = nB[2];
    state->B[3] = nB[3];
    state->B[4] = nB[4];
    state->B[5] = nB[5];
    state->B[6] = nB[6];
    state->B[7] = nB[7];
    state->B[8] = nB[8];
    state->B[9] = nB[9];
    state->B[10] = nB[10];

    state->L1 = nL1;
    state->R1 = nR1;
    state->L2 = nL2;
    state->R2 = nR2;
}

__device__ unsigned int sub_k2_shared(unsigned int in, const unsigned char *gf2_table, 
        const unsigned char *gf3_table,
        const unsigned char *d_s_box)
{
    unsigned char w0 = in & 0xFF;
    unsigned char w1 = (in >> 8) & 0xFF;
    unsigned char w2 = (in >> 16) & 0xFF;
    unsigned char w3 = (in >> 24) & 0xFF;

    unsigned char t0 = d_s_box[w0];
    unsigned char t1 = d_s_box[w1];
    unsigned char t2 = d_s_box[w2];
    unsigned char t3 = d_s_box[w3];

    unsigned char q0 = gf2_table[t0] ^ gf3_table[t1] ^ t2 ^ t3;
    unsigned char q1 = t0 ^ gf2_table[t1] ^ gf3_table[t2] ^ t3;
    unsigned char q2 = t0 ^ t1 ^ gf2_table[t2] ^ gf3_table[t3];
    unsigned char q3 = gf3_table[t0] ^ t1 ^ t2 ^ gf2_table[t3];

    return ((unsigned int)q3 << 24) | ((unsigned int)q2 << 16) | ((unsigned int)q1 << 8) | q0;
}


__device__ void setup_state_values_shared(const unsigned int *key, const unsigned int *iv, kcipher2_state *state, 
    const unsigned char *gf2_table, 
    const unsigned char *gf3_table,
    const unsigned char *d_s_box)
{
    unsigned int IK[12];
    IK[0] = key[0];
    IK[1] = key[1];
    IK[2] = key[2];
    IK[3] = key[3];

    IK[4] = IK[0] ^ sub_k2_shared((IK[3] << 8) ^ (IK[3] >> 24), gf2_table, gf3_table, d_s_box) ^ 0x01000000;
    IK[5] = IK[1] ^ IK[4];
    IK[6] = IK[2] ^ IK[5];
    IK[7] = IK[3] ^ IK[6];
    IK[8] = IK[4] ^ sub_k2_shared((IK[7] << 8) ^ (IK[7] >> 24), gf2_table, gf3_table, d_s_box) ^ 0x02000000;

    IK[9] = IK[5] ^ IK[8];
    IK[10] = IK[6] ^ IK[9];
    IK[11] = IK[7] ^ IK[10];

    state->A[0] = IK[4];
    state->A[1] = IK[3];
    state->A[2] = IK[2];
    state->A[3] = IK[1];
    state->A[4] = IK[0];

    state->B[0] = IK[10];
    state->B[1] = IK[11];
    state->B[2] = iv[0];
    state->B[3] = iv[1];
    state->B[4] = IK[8];
    state->B[5] = IK[9];
    state->B[6] = iv[2];
    state->B[7] = iv[3];
    state->B[8] = IK[7];
    state->B[9] = IK[5];
    state->B[10] = IK[6];

    state->L1 = state->R1 = state->L2 = state->R2 = 0x00000000;
}



__device__ void next_INIT_shared(kcipher2_state *state, 
    const unsigned char *gf2_table, 
    const unsigned char *gf3_table,
    const unsigned char *d_s_box,
    const unsigned int *d_amul0,
    const unsigned int *d_amul1,
    const unsigned int *d_amul2,
    const unsigned int *d_amul3)
{
    unsigned int temp2;
    unsigned int nL1 = sub_k2_shared(state->R2 + state->B[4], gf2_table, gf3_table, d_s_box);
    unsigned int nR1 = sub_k2_shared(state->L2 + state->B[9], gf2_table, gf3_table, d_s_box);
    unsigned int nL2 = sub_k2_shared(state->L1, gf2_table, gf3_table, d_s_box);
    unsigned int nR2 = sub_k2_shared(state->R1, gf2_table, gf3_table, d_s_box);

    unsigned int nA[5];
    nA[0] = state->A[1];
    nA[1] = state->A[2];
    nA[2] = state->A[3];
    nA[3] = state->A[4];

    unsigned int nB[11];
    nB[0] = state->B[1];
    nB[1] = state->B[2];
    nB[2] = state->B[3];
    nB[3] = state->B[4];
    nB[4] = state->B[5];
    nB[5] = state->B[6];
    nB[6] = state->B[7];
    nB[7] = state->B[8];
    nB[8] = state->B[9];
    nB[9] = state->B[10];

    unsigned int temp1 = (state->A[0] << 8) ^ d_amul0[(state->A[0] >> 24) & 0xFF];
    nA[4] = temp1 ^ state->A[3];
    nA[4] ^= nlf(state->B[0], state->R2, state->R1, state->A[4]);

    if (state->A[2] & 0x40000000)
        temp1 = (state->B[0] << 8) ^ d_amul1[(state->B[0] >> 24) & 0xFF];
    else
        temp1 = (state->B[0] << 8) ^ d_amul2[(state->B[0] >> 24) & 0xFF];

    // branchless version (not faster)
    //  unsigned int mask = -(unsigned int)(!!(state->A[2] & 0x40000000));
    //  int b0 = state->B[0] << 8;
    //  int b0_24 = (state->B[0] >> 24) & 0xFF;
    //  unsigned int candidate1 = (b0) ^ d_amul1[b0_24];
    //  unsigned int candidate2 = (b0) ^ d_amul2[b0_24];
    //  temp1 = candidate2 ^ ((candidate1 ^ candidate2) & mask);

    if (state->A[2] & 0x80000000)
        temp2 = (state->B[8] << 8) ^ d_amul3[(state->B[8] >> 24) & 0xFF];
    else
        temp2 = state->B[8];

    // branchless version (not faster)    
    // unsigned int mask2 = -(unsigned int)(!!(state->A[2] & 0x80000000));
    // unsigned int candidate3 = (state->B[8] << 8) ^ d_amul3[(state->B[8] >> 24) & 0xFF];
    // unsigned int candidate4 = state->B[8];
    // temp2 = candidate4 ^ ((candidate3 ^ candidate4) & mask2);

    nB[10] = temp1 ^ state->B[1] ^ state->B[6] ^ temp2;
    nB[10] ^= nlf(state->B[10], state->L2, state->L1, state->A[0]);

    state->A[0] = nA[0];
    state->A[1] = nA[1];
    state->A[2] = nA[2];
    state->A[3] = nA[3];
    state->A[4] = nA[4];

    state->B[0] = nB[0];
    state->B[1] = nB[1];
    state->B[2] = nB[2];
    state->B[3] = nB[3];
    state->B[4] = nB[4];
    state->B[5] = nB[5];
    state->B[6] = nB[6];
    state->B[7] = nB[7];
    state->B[8] = nB[8];
    state->B[9] = nB[9];
    state->B[10] = nB[10];

    state->L1 = nL1;
    state->R1 = nR1;
    state->L2 = nL2;
    state->R2 = nR2;
}


__device__ __forceinline__ unsigned long long bswap64(unsigned long long x)
{
           uint32_t hi = x >> 32;  // Upper 32 bits
           uint32_t lo = x & 0xFFFFFFFF;  // Lower 32 bits       
           // Swap bytes in each 32-bit part using __byte_perm
           hi = __byte_perm(hi, 0, 0x0123);
           lo = __byte_perm(lo, 0, 0x0123);
       
           // Swap the high and low parts and combine them back
           return ((uint64_t)lo << 32) | hi;           
}

__device__ unsigned long long kcipher2_encrypt_1_zero_block(const unsigned int *key, const unsigned int *iv)
{
    kcipher2_state state;
    setup_state_values(key, iv, &state);
    for (unsigned char i = 0; i < 24; i++)
        next_INIT(&state);

    unsigned int zh = nlf(state.B[10], state.L2, state.L1, state.A[0]);
    unsigned int zl = nlf(state.B[0], state.R2, state.R1, state.A[4]);
    return (((unsigned long long)zh) << 32) | zl;
}

__device__ unsigned long long kcipher2_encrypt_1_zero_block_shared(const unsigned int *key, const unsigned int *iv, 
        const unsigned char *gf2_table, 
        const unsigned char *gf3_table, 
        const unsigned char *d_s_box,
        const unsigned int *d_amul0,
        const unsigned int *d_amul1,
        const unsigned int *d_amul2,
        const unsigned int *d_amul3
    )
{
    kcipher2_state state;
    setup_state_values_shared(key, iv, &state, gf2_table, gf3_table, d_s_box);
    for (unsigned char i = 0; i < 24; i++)
        next_INIT_shared(&state, gf2_table, gf3_table, d_s_box, d_amul0, d_amul1, d_amul2, d_amul3);

    unsigned int zh = nlf(state.B[10], state.L2, state.L1, state.A[0]);
    unsigned int zl = nlf(state.B[0], state.R2, state.R1, state.A[4]);
    return (((unsigned long long)zh) << 32) | zl;
}

//---------------------------------------------------------------------------
// End of file
//---------------------------------------------------------------------------



// --- Multihash Kernel ---
//
// Each thread computes:
//   digest0 = SHA256(19-byte input)
//   v0 = digest0
//   for i=1..1499:
//     buffer = digest(i-1) || v0 || [i as 4-byte big-endian]
//     digest(i) = SHA256(buffer)
// aes256_encrypt(&key_ctx, counter, counter);
// Encrypt the (now updated) counter to produce the final output.
// aes256_encrypt(&key_ctx, counter, digest);

// this kernel only output 16 byte random number
// but we output 32 byte block, so its like this <16 byte random> <16 byte empty> <16 byte random> <16 byte empty> ...
// This will make it faster

__global__ void multihash_kernel(const uint8_t *input, uint8_t *output, size_t num)
{
    ssize_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num)
        return;


    __shared__  uint32_t sha256_k[64];
    __shared__ unsigned char gf2_table_shared[256];
    __shared__ uint8_t aes_sbox[256];
    __shared__  uint32_t aes_Rcon[7];

    if (threadIdx.x == 0) {
        memcpy(sha256_k, k, sizeof(sha256_k));
        memcpy(aes_sbox, sbox, sizeof(aes_sbox));
        memcpy(aes_Rcon, Rcon, sizeof(aes_Rcon));
        memcpy(gf2_table_shared, gf2_table, sizeof(gf2_table_shared));
    }
    __syncthreads(); // Make sure the data is loaded before use


    // Each input message is 19 bytes.
    const uint8_t *data = input + idx * 19;
    uint8_t digest[SHA256_DIGEST_SIZE];
    uint8_t v0[SHA256_DIGEST_SIZE];

    // First round: hash the 19-byte input.
    // sha256_hash(data, 19, digest);

    sha256_hash_19(data, digest, sha256_k);

    // Save initial digest as v0.
    for (int i = 0; i < SHA256_DIGEST_SIZE; i++)
    {
        v0[i] = digest[i];
    }
    uint8_t buffer[68]; // 32 bytes (digest) + 32 bytes (v0) + 4 bytes (counter)

    // Copy v0.
    for (int j = 0; j < SHA256_DIGEST_SIZE; j++)
    {
        buffer[32 + j] = v0[j];
    }

    // Write counter (big-endian).
    // buffer[64] = (i >> 24) & 0xff;
    // buffer[65] = (i >> 16) & 0xff;
    // max is 1500
    buffer[64] = 0;
    buffer[65] = 0;

    // 1,500 rounds total (first one is done above).
    for (int i = 1; i < 1500; i++)
    {
        // Copy current digest
	    #pragma unroll
        for (int j = 0; j < SHA256_DIGEST_SIZE; j++)
        {
            buffer[j] = digest[j];
        }

        buffer[66] = (i >> 8) & 0xff;
        buffer[67] = i & 0xff;

        // sha256_hash(buffer, 68, digest);
        sha256_hash_68(buffer, digest, sha256_k);
    }
#define AES_BLOCK_SIZE 16
    // -------- AES post-processing on final digest --------
    // Use final digest as the AES key.
    aes256_ctx key_ctx;
    aes256_set_encrypt_key(digest, &key_ctx, aes_sbox, aes_Rcon);
    uint8_t counter[16] = {0}; // clear counter

    // Encrypt the counter block.
    aes256_encrypt(&key_ctx, counter, counter, aes_sbox, gf2_table_shared);
    // Encrypt the (now updated) counter to produce the final output.
    aes256_encrypt(&key_ctx, counter, digest, aes_sbox, gf2_table_shared);

    // Write final (AES‑encrypted) digest to output.
    uint8_t *out = output + idx * SHA256_DIGEST_SIZE;

    uint32_t * out_u32 = (uint32_t *)out;
    uint32_t * digest_u32 = (uint32_t *)digest;

    //byte swap32 bit every 4 byte
    for (int i = 0; i < SHA256_DIGEST_SIZE / 4; i++)
    {
        out_u32[i] = __byte_perm(digest_u32[i], 0, 0x0123);
    }

}


__global__ void multihash_kernel_in_memory(uint8_t *output, uint64_t t_start, size_t num)
{
    ssize_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num)
        return;


    __shared__  uint32_t sha256_k[64];
    __shared__ unsigned char gf2_table_shared[256];
    __shared__ uint8_t aes_sbox[256];
    __shared__  uint32_t aes_Rcon[7];

    if (threadIdx.x == 0) {
        memcpy(sha256_k, k, sizeof(sha256_k));
        memcpy(aes_sbox, sbox, sizeof(aes_sbox));
        memcpy(aes_Rcon, Rcon, sizeof(aes_Rcon));
        memcpy(gf2_table_shared, gf2_table, sizeof(gf2_table_shared));
    }
    __syncthreads(); // Make sure the data is loaded before use
    uint64_t t = t_start + idx;

    // Each input message is 19 bytes.
    uint8_t data[19];
    for (int i =0; i < 19; i++) {
	    data[18-i]='0' + (t%10);
	    t /= 10;
    }
    
    uint8_t digest[SHA256_DIGEST_SIZE];
    // uint8_t v0[SHA256_DIGEST_SIZE];

    // First round: hash the 19-byte input.
    // sha256_hash(data, 19, digest);

    sha256_hash_19(data, digest, sha256_k);

    // Save initial digest as v0.
    // for (int i = 0; i < SHA256_DIGEST_SIZE; i++)
    // {
    //     v0[i] = digest[i];
    // }
    uint8_t buffer[68]; // 32 bytes (digest) + 32 bytes (v0) + 4 bytes (counter)

    // Copy v0.
    for (int j = 0; j < SHA256_DIGEST_SIZE; j++)
    {
        buffer[32 + j] = digest[j];
    }

    // Write counter (big-endian).
    // buffer[64] = (i >> 24) & 0xff;
    // buffer[65] = (i >> 16) & 0xff;
    // max is 1500
    buffer[64] = 0;
    buffer[65] = 0;

    // 1,500 rounds total (first one is done above).
    for (int i = 1; i < 1500; i++)
    {
        // Copy current digest.
        for (int j = 0; j < SHA256_DIGEST_SIZE; j++)
        {
            buffer[j] = digest[j];
        }

        buffer[66] = (i >> 8) & 0xff;
        buffer[67] = i & 0xff;

        // sha256_hash(buffer, 68, digest);
        sha256_hash_68(buffer, digest, sha256_k);
    }
#define AES_BLOCK_SIZE 16
    // -------- AES post-processing on final digest --------
    // Use final digest as the AES key.
    aes256_ctx key_ctx;
    aes256_set_encrypt_key(digest, &key_ctx, aes_sbox, aes_Rcon);
    uint8_t counter[16] = {0}; // clear counter

    // Encrypt the counter block.
    aes256_encrypt(&key_ctx, counter, counter, aes_sbox, gf2_table_shared);
    // Encrypt the (now updated) counter to produce the final output.
    aes256_encrypt(&key_ctx, counter, digest, aes_sbox, gf2_table_shared);

    // Write final (AES‑encrypted) digest to output.
    uint8_t *out = output + idx * SHA256_DIGEST_SIZE;

    uint32_t * out_u32 = (uint32_t *)out;
    uint32_t * digest_u32 = (uint32_t *)digest;

    //byte swap32 bit every 4 byte
    for (int i = 0; i < SHA256_DIGEST_SIZE / 4; i++)
    {
        out_u32[i] = __byte_perm(digest_u32[i], 0, 0x0123);
    }

}

//this is for chacha8 key, so no need to swap, and we nseed the 32 byte block
__global__ void multihash_kernel_in_memory_no_swap(uint8_t *output, uint64_t t_start, size_t num)
{
    ssize_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num)
        return;


    __shared__  uint32_t sha256_k[64];
    __shared__ unsigned char gf2_table_shared[256];
    __shared__ uint8_t aes_sbox[256];
    __shared__  uint32_t aes_Rcon[7];

    if (threadIdx.x == 0) {
        memcpy(sha256_k, k, sizeof(sha256_k));
        memcpy(aes_sbox, sbox, sizeof(aes_sbox));
        memcpy(aes_Rcon, Rcon, sizeof(aes_Rcon));
        memcpy(gf2_table_shared, gf2_table, sizeof(gf2_table_shared));
    }
    __syncthreads(); // Make sure the data is loaded before use
    uint64_t t = t_start + idx;

    // Each input message is 19 bytes.
    uint8_t data[19];
    for (int i =0; i < 19; i++) {
	    data[18-i]='0' + (t%10);
	    t /= 10;
    }
    
    uint8_t digest[SHA256_DIGEST_SIZE];
    // uint8_t v0[SHA256_DIGEST_SIZE];

    // First round: hash the 19-byte input.
    // sha256_hash(data, 19, digest);

    sha256_hash_19(data, digest, sha256_k);

    // Save initial digest as v0.
    // for (int i = 0; i < SHA256_DIGEST_SIZE; i++)
    // {
    //     v0[i] = digest[i];
    // }
    uint8_t buffer[68]; // 32 bytes (digest) + 32 bytes (v0) + 4 bytes (counter)

    // Copy v0.
    for (int j = 0; j < SHA256_DIGEST_SIZE; j++)
    {
        buffer[32 + j] = digest[j];
    }

    // Write counter (big-endian).
    // buffer[64] = (i >> 24) & 0xff;
    // buffer[65] = (i >> 16) & 0xff;
    // max is 1500
    buffer[64] = 0;
    buffer[65] = 0;

    // 1,500 rounds total (first one is done above).
    for (int i = 1; i < 1500; i++)
    {
        // Copy current digest.
        for (int j = 0; j < SHA256_DIGEST_SIZE; j++)
        {
            buffer[j] = digest[j];
        }

        buffer[66] = (i >> 8) & 0xff;
        buffer[67] = i & 0xff;

        // sha256_hash(buffer, 68, digest);
        sha256_hash_68(buffer, digest, sha256_k);
    }
#define AES_BLOCK_SIZE 16
    // -------- AES post-processing on final digest --------
    // Use final digest as the AES key.
    aes256_ctx key_ctx;
    aes256_set_encrypt_key(digest, &key_ctx, aes_sbox, aes_Rcon);
    uint8_t counter[16] = {0}; // clear counter

    // Encrypt the counter block.
    aes256_encrypt(&key_ctx, counter, counter, aes_sbox, gf2_table_shared);
    // Encrypt the (now updated) counter to produce the final output.
    aes256_encrypt(&key_ctx, counter, digest, aes_sbox, gf2_table_shared);

    // Write final (AES‑encrypted) digest to output.
    uint8_t *out = output + idx * SHA256_DIGEST_SIZE;

    #pragma unroll
    for (int i = 0; i < SHA256_DIGEST_SIZE; i++)
    {
        out[i] = digest[i];
    }
}



//chacha8 key doesn't need swapping
__global__ void multihash_kernel_noswap(const uint8_t *input, uint8_t *output, size_t num)
{
    ssize_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num)
        return;


    __shared__  uint32_t sha256_k[64];
    __shared__ unsigned char gf2_table_shared[256];
    __shared__ uint8_t aes_sbox[256];
    __shared__  uint32_t aes_Rcon[7];

    if (threadIdx.x == 0) {
        memcpy(sha256_k, k, sizeof(sha256_k));
        memcpy(aes_sbox, sbox, sizeof(aes_sbox));
        memcpy(aes_Rcon, Rcon, sizeof(aes_Rcon));
        memcpy(gf2_table_shared, gf2_table, sizeof(gf2_table_shared));
    }
    __syncthreads(); // Make sure the data is loaded before use


    // Each input message is 19 bytes.
    const uint8_t *data = input + idx * 19;
    uint8_t digest[SHA256_DIGEST_SIZE];
    uint8_t v0[SHA256_DIGEST_SIZE];

    // First round: hash the 19-byte input.
    // sha256_hash(data, 19, digest);

    sha256_hash_19(data, digest, sha256_k);

    // Save initial digest as v0.
    for (int i = 0; i < SHA256_DIGEST_SIZE; i++)
    {
        v0[i] = digest[i];
    }
    uint8_t buffer[68]; // 32 bytes (digest) + 32 bytes (v0) + 4 bytes (counter)

    // Copy v0.
    for (int j = 0; j < SHA256_DIGEST_SIZE; j++)
    {
        buffer[32 + j] = v0[j];
    }

    // Write counter (big-endian).
    // buffer[64] = (i >> 24) & 0xff;
    // buffer[65] = (i >> 16) & 0xff;
    // max is 1500
    buffer[64] = 0;
    buffer[65] = 0;

    // 1,500 rounds total (first one is done above).
    for (int i = 1; i < 1500; i++)
    {
        // Copy current digest.
        for (int j = 0; j < SHA256_DIGEST_SIZE; j++)
        {
            buffer[j] = digest[j];
        }

        buffer[66] = (i >> 8) & 0xff;
        buffer[67] = i & 0xff;

        // sha256_hash(buffer, 68, digest);
        sha256_hash_68(buffer, digest, sha256_k);
    }
#define AES_BLOCK_SIZE 16
    // -------- AES post-processing on final digest --------
    // Use final digest as the AES key.
    aes256_ctx key_ctx;
    aes256_set_encrypt_key(digest, &key_ctx, aes_sbox, aes_Rcon);
    uint8_t counter[16] = {0}; // clear counter

    // Encrypt the counter block.
    aes256_encrypt(&key_ctx, counter, counter, aes_sbox, gf2_table_shared);
    // Encrypt the (now updated) counter to produce the final output.
    aes256_encrypt(&key_ctx, counter, digest, aes_sbox, gf2_table_shared);

    // Write final (AES‑encrypted) digest to output.
    uint8_t *out = output + idx * SHA256_DIGEST_SIZE;

     #pragma unroll
    for (int i = 0; i < SHA256_DIGEST_SIZE; i++)
    {
        out[i] = digest[i];
    }
}



uint64_t get_time_in_nanosecond()
{
    struct timespec time;
    clock_gettime(CLOCK_MONOTONIC, &time);
    return (uint64_t)time.tv_sec * 1000000000 + (uint64_t)time.tv_nsec;
}

#define TEST_ENCRYPT_BLOCK_SIZE 32

// Test kernel: each thread processes one key/IV pair.
// this is to test raw encryption speed
__global__ void test_kcipher2_kernel(const uint8_t *in, uint8_t *out, size_t N)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N)
        return;

    // Each input block: first 16 bytes = Key, next 16 bytes = IV.
    const int in_block_size = 32;                       // 16-byte key + 16-byte IV
    const int out_block_size = TEST_ENCRYPT_BLOCK_SIZE; // 128-byte ciphertext

    // Pointers to the key/IV for this thread.
    const uint8_t *block_in = in + idx * in_block_size;

    // Load key (16 bytes) and IV (16 bytes) into local arrays.
    unsigned int key_local[4];
    unsigned int iv_local[4];
    for (int i = 0; i < 4; i++)
    {
        int base_key = i * 4;
        key_local[i] = ((unsigned int)block_in[base_key] << 24) |
                       ((unsigned int)block_in[base_key + 1] << 16) |
                       ((unsigned int)block_in[base_key + 2] << 8) |
                       ((unsigned int)block_in[base_key + 3]);
    }
    for (int i = 0; i < 4; i++)
    {
        int base_iv = 16 + i * 4;
        iv_local[i] = ((unsigned int)block_in[base_iv] << 24) |
                      ((unsigned int)block_in[base_iv + 1] << 16) |
                      ((unsigned int)block_in[base_iv + 2] << 8) |
                      ((unsigned int)block_in[base_iv + 3]);
    }

    unsigned char ciphertext[8];

    long long res = kcipher2_encrypt_1_zero_block(key_local, iv_local);
    // copy res to ciphertext
    for (int i = 0; i < 8; i++)
    {
        ciphertext[i] = (res >> (56 - i * 8)) & 0xFF;
    }

    // Write the output ciphertext.
    size_t out_offset = idx * out_block_size;
    for (int i = 0; i < 8; i++)
    {
        out[out_offset + i] = ciphertext[i];
    }
}

void fill_input(uint8_t *h_input, uint64_t start, size_t num)
{
    char buffer[20];
    snprintf(buffer, 20, "%019lu", start);
    for (size_t i = 0; i < num; i++)
    {
        // copy buffer to input
        memcpy(h_input + i * 19, buffer, 19);
        // increment buffer, starting from last index, until it reaches '9'
        for (int j = 18; j >= 0; j--)
        {
            if (buffer[j] == '9')
            {
                buffer[j] = '0';
            }
            else
            {
                buffer[j]++;
                break;
            }
        }
    }
}

int test_encryption_only(size_t num)
{
    printf("Count %zu \n", num);
    size_t input_size = num * TEST_ENCRYPT_BLOCK_SIZE;

    // Query device properties
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, gpuIndex);

    int maxThreadsPerBlock = prop.maxThreadsPerBlock; // Maximum allowed per block
    int blockSize = maxThreadsPerBlock;               // Dynamically set blockSize
    // int blockSize = 256;
    int gridSize = (num + blockSize - 1) / blockSize; // Ensure full coverage

    printf("Using blockSize = %d, gridSize = %d\n", blockSize, gridSize);

    size_t output_size = num * TEST_ENCRYPT_BLOCK_SIZE;

    printf("GPU needed memory: %.2f MB\n", (input_size + output_size) / 1024.0 / 1024.0);

    uint8_t *h_input = (uint8_t *)malloc(input_size);
    // test the blank/zero encryption
    // memset(h_input, 0, input_size);
    // fill with random
    fill_input(h_input, TEST_TIMESTAMP, num * (32 / 19));

    uint8_t *h_output_enc = (uint8_t *)malloc(output_size);

    // Allocate device memory.
    uint8_t *d_input, *d_output_enc;

    printf("Allocating device memory...%.2f Mb\n", (input_size + output_size) / 1024.0 / 1024.0);

    if (hipMalloc(&d_input, input_size) != hipSuccess)
    {
        fprintf(stderr, "Error: hipMalloc failed for d_input\n");
        return 1;
    }
    if (hipMalloc(&d_output_enc, output_size) != hipSuccess)
    {
        fprintf(stderr, "Error: hipMalloc failed for d_output_enc\n");
        return 1;
    }
    uint64_t start = get_time_in_nanosecond();

    // copy to device
    hipMemcpy(d_input, h_input, input_size, hipMemcpyHostToDevice);

    test_kcipher2_kernel<<<gridSize, blockSize>>>(d_input, d_output_enc, num);
    hipDeviceSynchronize();

    // debug encryption output
    hipMemcpy(h_output_enc, d_output_enc, output_size, hipMemcpyDeviceToHost);

    uint64_t end = get_time_in_nanosecond();

    printf("Enc Time: %f ms\n", (end - start) / 1000000.0);

    // print speed
    printf("Enc Speed: %f enc per second\n", num / ((end - start) / 1000000000.0));

    // print first plaintext
    printf("Plaintext for first input: ");
    for (size_t i = 0; i < 19; i++)
    {
        printf("%c", h_input[i]);
    }
    printf("\n");

    // For demonstration, print the first output digest in hexadecimal.
    printf("Encrypted for first input: ");
    for (size_t i = 0; i < TEST_ENCRYPT_BLOCK_SIZE; i++)
    {
        printf("%02x", h_output_enc[i]);
    }
    printf("\n");
    // print last
    // print plaintext
    printf("Plaintext for last input: ");
    for (size_t i = 19 * num - 19; i < 19 * num; i++)
    {
        printf("%c", h_input[i]);
    }
    printf("\n");
    printf("Encrypted for last input: offs <%zu>: ", num * TEST_ENCRYPT_BLOCK_SIZE - TEST_ENCRYPT_BLOCK_SIZE);
    for (size_t i = TEST_ENCRYPT_BLOCK_SIZE * num - TEST_ENCRYPT_BLOCK_SIZE; i < TEST_ENCRYPT_BLOCK_SIZE * num; i++)
    {
        printf("%02x", h_output_enc[i]);
    }
    printf("\n");

    // Cleanup.
    hipFree(d_input);
    hipFree(d_output_enc);
    free(h_input);
    free(h_output_enc);

    return 0;
}

#define DIGEST_SIZE 32
#define KEY_IV_SIZE 16 // we use first 16 bytes for key/iv
#define KCIPHER_OUT_SIZE 8

// KCIPHER2, single kernel

// --- Host Code ---

//for testing only, it will save all random number generated from a given second

int save_random(uint64_t start_time, const char *filename)
{
    printf("Saving all seeds for %lu to %s\n", start_time, filename);

    uint64_t timer_start = get_time_in_nanosecond();

    //do it for every 10 million
    size_t num = 10*1000*1000;
    size_t input_size = num * 19 * sizeof(uint8_t);
    uint8_t *h_input = (uint8_t *)malloc(input_size);
    size_t output_size = num * SHA256_DIGEST_SIZE * sizeof(uint8_t);
    uint8_t *h_output = (uint8_t *)malloc(output_size);

    uint8_t *h_output_non_zeroes = (uint8_t *)malloc(num * 16);

    // Allocate device memory.
    uint8_t *d_input, *d_output;
    if (hipMalloc(&d_input, input_size) != hipSuccess)
    {
        fprintf(stderr, "Error: hipMalloc failed for d_input\n");
        return 1;
    }
    if (hipMalloc(&d_output, output_size) != hipSuccess)
    {
        fprintf(stderr, "Error: hipMalloc failed for d_output\n");
        return 1;
    }

    int blockSize = 256;
    int gridSize = (num + blockSize - 1) / blockSize; // Ensure full coverage
    printf("Using blockSize = %d, gridSize = %d\n", blockSize, gridSize);

    FILE *f = fopen(filename, "wb");
    if (f == NULL)
    {
        fprintf(stderr, "Error: Cannot open file %s\n", filename);
        return 1;
    }

    for (size_t i = 0; i < 100; i++) {
        printf("Saving %zu %%\r", i);fflush(stdout);
        fill_input(h_input, start_time + i * num, num); 
        //copy to CUDA
        hipMemcpy(d_input, h_input, input_size, hipMemcpyHostToDevice);

        multihash_kernel<<<gridSize, blockSize>>>(d_input, d_output, num);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
        {
            fprintf(stderr, "Error: %s\n", hipGetErrorString(err));
            return 1;
        }
        hipDeviceSynchronize();        
        hipMemcpy(h_output, d_output, output_size, hipMemcpyDeviceToHost);
        //copy every 16 bytes
        for (size_t j = 0; j < num; j++)
        {
            memcpy(h_output_non_zeroes + j * 16, h_output + j * DIGEST_SIZE, 16);
        }
        fwrite(h_output_non_zeroes, 16, num, f);

    }
    fclose(f);
    uint64_t timer_end = get_time_in_nanosecond();

    printf("DONE saved to %s: total time %.2f second \n", filename, (timer_end - timer_start) / 1000000000.0);
    
    return 0;
}

int test_generate_random_only_in_gpu(size_t num)
{
    uint64_t t_start = TEST_TIMESTAMP + 2000; // TEST:  0 is for chacha, +1000 for chacha_nonce	

    size_t output_size = num * SHA256_DIGEST_SIZE * sizeof(uint8_t);
    printf("Output size: %.2f MB\n", output_size / 1024.0 / 1024.0);
    uint8_t *h_output = (uint8_t *)malloc(output_size);

    uint8_t *d_output;
    if (hipMalloc(&d_output, output_size) != hipSuccess)
    {
        fprintf(stderr, "Error: hipMalloc failed for d_output\n");
        return 1;
    }
    int blockSize = 256;
    int gridSize = (num + blockSize - 1) / blockSize; // Ensure full coverage
    uint64_t start = get_time_in_nanosecond();
    multihash_kernel_in_memory<<<gridSize, blockSize>>>(d_output, t_start, num);
    hipDeviceSynchronize();
    uint64_t end = get_time_in_nanosecond();    
    hipMemcpy(h_output, d_output, output_size, hipMemcpyDeviceToHost);

    // print first and last
    printf("First random ");
    for (int i = 0; i < DIGEST_SIZE; i++)
    {
        printf("%02x", h_output[i]);
    }
    printf("\n");
    printf("Last random ");
    for (size_t i = DIGEST_SIZE * num - DIGEST_SIZE; i < DIGEST_SIZE * num; i++)
    {
        printf("%02x", h_output[i]);
    }
    printf("\n");

    printf("Total Time: %f ms\n", (end - start) / 1000000.0);
    printf("Speed: %f hashes per second\n", num / ((end - start) / 1000000000.0));
    
    
    return 0;    
}

int test_generate_random_only(size_t num)
{
    printf("Test generate random only: %lu\n", num);
    size_t input_size = num * 19 * sizeof(uint8_t);
    printf("Input size: %.2f MB\n", input_size / 1024.0 / 1024.0);

    // Allocate host memory.
    uint8_t *h_input = (uint8_t *)malloc(input_size);

    uint64_t t_start = TEST_TIMESTAMP + 2000; // TEST:  0 is for chacha, +1000 for chacha_nonce

    // output
    size_t output_size = num * SHA256_DIGEST_SIZE * sizeof(uint8_t);
    printf("Output size: %.2f MB\n", output_size / 1024.0 / 1024.0);

    uint8_t *h_output = (uint8_t *)malloc(output_size);

    uint64_t start_fill = get_time_in_nanosecond();

    fill_input(h_input, t_start, num);
    uint64_t end_fill = get_time_in_nanosecond();
    printf("Fill Time: %f ms\n", (end_fill - start_fill) / 1000000.0);

    // Allocate device memory.
    uint8_t *d_input, *d_output;
    if (hipMalloc(&d_input, input_size) != hipSuccess)
    {
        fprintf(stderr, "Error: hipMalloc failed for d_input\n");
        return 1;
    }
    if (hipMalloc(&d_output, output_size) != hipSuccess)
    {
        fprintf(stderr, "Error: hipMalloc failed for d_output\n");
        return 1;
    }

    uint64_t start = get_time_in_nanosecond();

    hipMemcpy(d_input, h_input, input_size, hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (num + blockSize - 1) / blockSize; // Ensure full coverage

    printf("Using blockSize = %d, gridSize = %d\n", blockSize, gridSize);

    printf("Starting timestamp -> random calculation...\n");
    fflush(stdout);

    multihash_kernel<<<gridSize, blockSize>>>(d_input, d_output, num);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: %s\n", hipGetErrorString(err));
        return 1;
    }

    hipDeviceSynchronize();
    uint64_t end1 = get_time_in_nanosecond();
    hipMemcpy(h_output, d_output, output_size, hipMemcpyDeviceToHost);

    uint64_t end = get_time_in_nanosecond();

    // print first and last
    printf("First random ");
    for (int i = 0; i < DIGEST_SIZE; i++)
    {
        printf("%02x", h_output[i]);
    }
    printf("\n");
    printf("Last random ");
    for (size_t i = DIGEST_SIZE * num - DIGEST_SIZE; i < DIGEST_SIZE * num; i++)
    {
        printf("%02x", h_output[i]);
    }
    printf("\n");

    printf("Total Time: %f ms\n", (end - start) / 1000000.0);
    // print speed per second
    printf("Speed: %f hashes per second\n", num / ((end1 - start) / 1000000000.0));
    // Cleanup.
    hipFree(d_input);
    hipFree(d_output);
    free(h_input);
    free(h_output);
    return 0;
}


#define U8TO32_LITTLE(p) (*(const uint32_t *)(p))

typedef struct {
    uint32_t a, b, c, d, e, f, g, h, i, j, k, l, m, n, o, p;
} BLOCK;


// Device helper: rotate left 32-bit value
__device__ __forceinline__ uint32_t _rotl(uint32_t x, int n) {
    return (x << n) | (x >> (32 - n));
}


#define QROUND(a, b, c, d)       \
    d = _rotl(d ^ (a += b), 16); \
    b = _rotl(b ^ (c += d), 12); \
    d = _rotl(d ^ (a += b), 8);  \
    b = _rotl(b ^ (c += d), 7)
#define FROUND                  \
    QROUND(x.d, x.h, x.l, x.p); \
    QROUND(x.c, x.g, x.k, x.o); \
    QROUND(x.b, x.f, x.j, x.n); \
    QROUND(x.a, x.e, x.i, x.m); \
    QROUND(x.a, x.f, x.k, x.p); \
    QROUND(x.b, x.g, x.l, x.m); \
    QROUND(x.c, x.h, x.i, x.n); \
    QROUND(x.d, x.e, x.j, x.o)
#define FFINAL  \
    x.a += j.a; \
    x.b += j.b; \
    x.c += j.c; \
    x.d += j.d; \
    x.e += j.e; \
    x.f += j.f; \
    x.g += j.g; \
    x.h += j.h; \
    x.i += j.i; \
    x.j += j.j; \
    x.k += j.k; \
    x.l += j.l; \
    x.m += j.m; \
    x.n += j.n; \
    x.o += j.o; \
    x.p += j.p

//in contains key/nonce
//offset is the match offset
//out_flag is the flag: ts << 32 | offset
__global__ void chacha8_encrypt_and_match(const uint8_t *in, size_t offset,
    unsigned long long *out_flag,
    const unsigned long long value,
    int N)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= N)
    {
        return;
    }

    size_t idx_offs = idx + offset;

    if (idx_offs >= N)
    {
        return;
    }

    const int in_block_size = 32;

    // Pointers to the key/IV for this thread.
    const uint8_t *block_in = in + idx * in_block_size;
    const uint8_t *block_offset_in = in + idx_offs * in_block_size;

    const uint8_t *k = block_in;
    const uint8_t *iv = block_offset_in;
    const char constants[16] = {'e', 'x', 'p', 'a', 'n', 'd', ' ', '1', '6', '-', 'b', 'y', 't', 'e', ' ', 'k'};


    uint32_t state[16];
    state[4] = U8TO32_LITTLE(k + 0);
    state[5] = U8TO32_LITTLE(k + 4);
    state[6] = U8TO32_LITTLE(k + 8);
    state[7] = U8TO32_LITTLE(k + 12);
    state[8] = U8TO32_LITTLE(k + 0);
    state[9] = U8TO32_LITTLE(k + 4);
    state[10] = U8TO32_LITTLE(k + 8);
    state[11] = U8TO32_LITTLE(k + 12);
    state[0] = U8TO32_LITTLE(constants + 0);
    state[1] = U8TO32_LITTLE(constants + 4);
    state[2] = U8TO32_LITTLE(constants + 8);
    state[3] = U8TO32_LITTLE(constants + 12);
    state[12] = 0;
    state[13] = 0;
    state[14] = U8TO32_LITTLE(iv + 0);
    state[15] = U8TO32_LITTLE(iv + 4);

    BLOCK x;
    BLOCK j;

    memcpy(&j, state, sizeof(BLOCK)); //j is for final addition
    j.m = 0;
    j.n = 0;

    memcpy(&x, &j, sizeof(BLOCK)); //FROUND will modify x

    FROUND;
    FROUND;
    FROUND;
    FROUND;
    FFINAL;

    uint64_t *result = (uint64_t *)&x;
    if (*result == value) {
        unsigned long long encoded_offset_and_index = (unsigned long long)offset << 32 | idx;
        *out_flag = encoded_offset_and_index;
    }

}

__global__ void encrypt_and_search_offset(const uint8_t *in, size_t offset,
                                          unsigned long long *out_flag,
                                          const unsigned long long *masks,
                                          const unsigned long long *values,
                                          int numComb,
                                          int N)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= N)
    {
        return;
    }

    size_t idx_offs = idx + offset;

    if (idx_offs >= N)
    {
        return;
    }


    __shared__ unsigned char gf2_table_shared[256];
    __shared__ unsigned char gf3_table_shared[256];
    __shared__ unsigned char d_s_box_shared[256];
    __shared__ unsigned int d_amul0_shared[256];
    __shared__ unsigned int d_amul1_shared[256];
    __shared__ unsigned int d_amul2_shared[256];
    __shared__ unsigned int d_amul3_shared[256];
    __shared__ unsigned long long shared_masks[MAX_MATCHES];
    __shared__ unsigned long long shared_values[MAX_MATCHES];

    if (threadIdx.x == 0) {
        //copy gf2_table from const
        memcpy(gf2_table_shared, gf2_table, 256);
        memcpy(gf3_table_shared, gf3_table, 256);
        memcpy(d_s_box_shared, d_s_box, 256);
        memcpy(d_amul0_shared, d_amul0, 256 * sizeof(int));
        memcpy(d_amul1_shared, d_amul1, 256 * sizeof(int));
        memcpy(d_amul2_shared, d_amul2, 256 * sizeof(int));
        memcpy(d_amul3_shared, d_amul3, 256 * sizeof(int));
        for (int i = 0; i < numComb; i++)
        {
            shared_masks[i] = masks[i];
            shared_values[i] = values[i];
        }
    }
    __syncthreads(); // Make sure the data is loaded before use
    

    const int in_block_size = 32;

    // Pointers to the key/IV for this thread.
    const uint8_t *block_in = in + idx * in_block_size;
    const uint8_t *block_offset_in = in + idx_offs * in_block_size;

    // Load key (16 bytes) and IV (16 bytes) into local arrays.
    unsigned int key_local[4];
    unsigned int iv_local[4];
    const uint8_t *key_in = block_in;


    const uint8_t *iv_in = block_offset_in;

    memcpy(key_local, key_in, 16);


    memcpy(iv_local, iv_in, 16);

    //long long res = kcipher2_encrypt_1_zero_block(key_local, iv_local);
    long long res = kcipher2_encrypt_1_zero_block_shared(key_local, iv_local, gf2_table_shared, gf3_table_shared, d_s_box_shared,
        d_amul0_shared, d_amul1_shared, d_amul2_shared, d_amul3_shared);


    // compare with matches
    unsigned long long in_val = bswap64(res);

    unsigned int flag = 0;

    for (int i = 0; i < numComb; i++) {
        // diff is 0 if the masked input equals the expected value
        unsigned long long diff = (in_val & shared_masks[i]) ^ shared_values[i];
        // Compute a branch-free match: returns nonzero if diff is zero.
        flag |= (1 - ((diff | -diff) >> 63));   
    }
    if (flag) {
        //we found it, now find out exactly which one did we find
        for (int i = 0; i < numComb; i++) {
            if ((in_val & shared_masks[i]) == shared_values[i]) {
                //idx: max 1 billion, use 30 bits
                //offset: max 64 million, use 26
                //numComb: max 128, use 7
                unsigned long long encoded_offset_and_index = (unsigned long long)(idx << 34 | offset << 8 | i << 1 | 1);

                *out_flag = encoded_offset_and_index;

                break;
            }
        }
    }

}

void decode_offset_and_index(unsigned long long encoded_offset_and_index, 
                        size_t *offset, size_t *index, size_t *matchPos)
{
    //idx: max 1 billion, use 30 bits
    //offset: max 64 million, use 26
    //matchPos: max 128, use 7 bits
    //final 1 bit to ensure we have true value    
    //how it was encoded:
    //unsigned long long encoded_offset_and_index = (unsigned long long)(idx << 34 | offset << 8 | i << 1 | 1);                
    //decode:
    *index = (encoded_offset_and_index >> 34) & 0x3fffffff;
    *offset = (encoded_offset_and_index >> 8) & 0x3ffffff;
    *matchPos = (encoded_offset_and_index >> 1) & 0x7f;
}

__global__ void encrypt_and_search(const uint8_t *in, uint8_t *out, uint32_t k1, uint32_t k2, uint32_t k3, uint32_t k4,
                                   int *out_flag,
                                   const unsigned long long *masks,
                                   const unsigned long long *values,
                                   int numComb,
                                   int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N)
        return;

    // for (int i =0; i < 16; i++) {
    //     printf("%02x", in[i]);
    // }
    // printf("\n");

    // Each input block: first 16 bytes = Key, next 16 bytes = IV.
    const int in_block_size = 32;                       // 16-byte IV + zeroes
    const int out_block_size = TEST_ENCRYPT_BLOCK_SIZE; // block size

    // Pointers to the key/IV for this thread.
    const uint8_t *block_in = in + idx * in_block_size;

    // Load key (16 bytes) and IV (16 bytes) into local arrays.
    __shared__ unsigned int key_local[4];

    __shared__ unsigned char gf2_table_shared[256];
    __shared__ unsigned char gf3_table_shared[256];
    __shared__ unsigned char d_s_box_shared[256];
    __shared__ unsigned int d_amul0_shared[256];
    __shared__ unsigned int d_amul1_shared[256];
    __shared__ unsigned int d_amul2_shared[256];
    __shared__ unsigned int d_amul3_shared[256];
    __shared__ unsigned long long shared_masks[MAX_MATCHES];
    __shared__ unsigned long long shared_values[MAX_MATCHES];

    unsigned int iv_local[4];
    if (threadIdx.x == 0) {
        key_local[0] = k1;
        key_local[1] = k2;
        key_local[2] = k3;
        key_local[3] = k4;
        //copy gf2_table from const
        memcpy(gf2_table_shared, gf2_table, 256);
        memcpy(gf3_table_shared, gf3_table, 256);
        memcpy(d_s_box_shared, d_s_box, 256);
        memcpy(d_amul0_shared, d_amul0, 256 * sizeof(int));
        memcpy(d_amul1_shared, d_amul1, 256 * sizeof(int));
        memcpy(d_amul2_shared, d_amul2, 256 * sizeof(int));
        memcpy(d_amul3_shared, d_amul3, 256 * sizeof(int));
        for (int i = 0; i < numComb; i++)
        {
            shared_masks[i] = masks[i];
            shared_values[i] = values[i];
        }
    }
    __syncthreads(); // Make sure the data is loaded before use

    const uint8_t *iv_in = block_in;

    
    memcpy(iv_local, iv_in,16);

    //long long res = kcipher2_encrypt_1_zero_block(key_local, iv_local);
    long long res = kcipher2_encrypt_1_zero_block_shared(key_local, iv_local, gf2_table_shared, gf3_table_shared, d_s_box_shared,
        d_amul0_shared, d_amul1_shared, d_amul2_shared, d_amul3_shared);

    // compare with matches
    unsigned long long in_val = bswap64(res);
    unsigned int flag = 0;
    for (int i = 0; i < numComb; i++)
    {
        if ((in_val & masks[i]) == values[i])
        {
            *out_flag = 1;
            flag = 1;
            break;
        }
    }


#if 1
#ifdef NUM_MATCHES    
    #pragma unroll
    for (int i = 0; i < NUM_MATCHES; i++) {
#else
    for (int i = 0; i < numComb; i++) {
#endif        

        // diff is 0 if the masked input equals the expected value
        unsigned long long diff = (in_val & shared_masks[i]) ^ shared_values[i];
        // Compute a branch-free match: returns nonzero if diff is zero.
        flag |= (1 - ((diff | -diff) >> 63));   
    }
    if (flag)
        *out_flag = flag;

#endif
        
    if (flag)
    {
        // Write the output ciphertext.
        int out_offset = idx * out_block_size;
#pragma unroll
        for (int i = 0; i < 8; i++) // only copy changed
        {
            out[out_offset + i] = (res >> (56 - i * 8)) & 0xFF;
        }
    }
}



int do_bruteforce_offset(const char *filename)
{
    printf("Bruteforce Offset\n");
    json config;

    std::string json_file = filename;

    std::string checkpoint = json_file + ".checkpoint.json";

    std::ifstream f(filename);
    if (!f.is_open())
    {
        fprintf(stderr, "Error: Could not open config file %s\n", filename);
        return 1;
    }

    try
    {
        f >> config;
    }
    catch (json::parse_error &e)
    {
        fprintf(stderr, "Error parsing JSON: %s\n", e.what());
        return 1;
    }

    size_t *offsets = 0;
    size_t offset_count = 0;
    //open offset.txt
    FILE *f_offset = fopen("offset.txt", "r");
    if (f_offset == NULL)
    {
        fprintf(stderr, "Error: Could not open offset file\n");
        return 1;
    }
    //read line by line, convert to integer
    char line[256];
    while (fgets(line, sizeof(line), f_offset))
    {
        offsets = (size_t *)realloc(offsets, (offset_count + 1) * sizeof(size_t));
        offsets[offset_count] = strtoull(line, NULL, 10);
        offset_count++;
    }
    fclose(f_offset);
    printf("Offset count: %zu\n", offset_count);
    //print first and last offset
    printf("First offset: %zu\n", offsets[0]);
    printf("Last offset: %zu\n", offsets[offset_count - 1]);


    size_t num;
    uint64_t t_start;
    //size_t enc_count;
    size_t offset;

    size_t matches_size = 0;
    uint64_t *matches = 0;
    uint64_t *masks = 0;

    try
    {
        t_start = config["start_timestamp"].get<uint64_t>(); // start T3

        num = config["count"].get<size_t>(); // stop at T3 + count ns

        offset = config["offset"].get<size_t>(); // we start at T3 + offset ns

        //enc_count = config["brute_force_time_range"].get<size_t>(); // we stop at T3 + offset + enc_count ns

        // "mathes": [
        //     {
        //         "plaintext": "0x00000000",
        //         "encrypted": "0x00000000",
        //         "bitmask": "0xffffffff"
        //     },
        //     {
        //         "plaintext": "0x00000000",
        //         "encrypted": "0x00000001",
        //         "bitmask": "0xffffffff"
        //     }
        // ]
        // parse matches, and put it in array (match = plaintext^encrypted)
        if (config.contains("matches"))
        {
            matches_size = config["matches"].size();
            assert(matches_size < MAX_MATCHES);
            assert(matches_size > 0);
            matches = (uint64_t *)malloc(matches_size * sizeof(uint64_t));
            masks = (uint64_t *)malloc(matches_size * sizeof(uint64_t));
            for (size_t i = 0; i < matches_size; i++)
            {
                uint64_t plaintext = std::stoull(config["matches"][i]["plaintext"].get<std::string>(), 0, 16);
                uint64_t encrypted = std::stoull(config["matches"][i]["encrypted"].get<std::string>(), 0, 16);
                uint64_t bitmask = std::stoull(config["matches"][i]["bitmask"].get<std::string>(), 0, 16);
                matches[i] = plaintext ^ encrypted;
                masks[i] = bitmask;
                printf("Match %zu: %016lx bitmask %016lx \n", i, matches[i], masks[i]);
            }
            
        }
    }
    catch (json::exception &e)
    {
        fprintf(stderr, "Error reading JSON values: %s\n", e.what());
        return 1;
    }

    printf("Configuration:\n");
    printf("num: %zu\n", num);
    printf("t_start: %lu\n", t_start);
    //printf("enc_count: %zu\n", enc_count);
    printf("offset: %zu\n", offset);

    //printf("Brute forcing: %zu enc count %zu\n", num, enc_count);
    size_t input_size = num * 19 * sizeof(uint8_t);
    printf("Input size: %.2f MB\n", input_size / 1024.0 / 1024.0);

    // Allocate host memory.
    uint8_t *h_input = (uint8_t *)malloc(input_size);

    // output
    size_t output_size = num * SHA256_DIGEST_SIZE * sizeof(uint8_t);
    printf("Output size: %.2f MB\n", output_size / 1024.0 / 1024.0);

    uint8_t *h_output = (uint8_t *)malloc(output_size);

    //size_t enc_output_size = enc_count * TEST_ENCRYPT_BLOCK_SIZE * sizeof(uint8_t);

    //uint8_t *h_output_enc = (uint8_t *)malloc(enc_output_size);

    uint64_t start_fill = get_time_in_nanosecond();

    fill_input(h_input, t_start, num); // GENERATE TS

    uint64_t end_fill = get_time_in_nanosecond();
    printf("Fill Time: %f ms\n", (end_fill - start_fill) / 1000000.0);

    // Allocate device memory.
    uint8_t *d_input, *d_output;
    //uint8_t *d_output_enc;
    unsigned long long *d_found;
    unsigned long long *d_matches;
    unsigned long long *d_masks;

    if (hipMalloc(&d_input, input_size) != hipSuccess)
    {
        fprintf(stderr, "Error: hipMalloc failed for d_input\n");
        return 1;
    }
    if (hipMalloc(&d_output, output_size) != hipSuccess)
    {
        fprintf(stderr, "Error: hipMalloc failed for d_output\n");
        return 1;
    }
    // if (hipMalloc(&d_output_enc, enc_output_size) != hipSuccess)
    // {
    //     fprintf(stderr, "Error: hipMalloc failed for d_output_enc\n");
    //     return 1;
    // }
    if (hipMalloc(&d_found, sizeof(unsigned long long)) != hipSuccess)
    {
        fprintf(stderr, "Error: hipMalloc failed for d_found\n");
        return 1;
    }
    if (hipMalloc(&d_matches, matches_size * sizeof(uint64_t)) != hipSuccess)
    {
        fprintf(stderr, "Error: hipMalloc failed for d_matches\n");
        return 1;
    }
    if (hipMalloc(&d_masks, matches_size * sizeof(uint64_t)) != hipSuccess)
    {
        fprintf(stderr, "Error: hipMalloc failed for d_masks\n");
        return 1;
    }

    unsigned long long zero = 0;
    hipMemcpy(d_found, &zero, sizeof(unsigned long long), hipMemcpyHostToDevice);

    // copy matches and masks
    hipMemcpy(d_matches, matches, matches_size * sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(d_masks, masks, matches_size * sizeof(uint64_t), hipMemcpyHostToDevice);

    uint64_t start = get_time_in_nanosecond();

    hipMemcpy(d_input, h_input, input_size, hipMemcpyHostToDevice);

    // Launch kernel.
    int blockSize = 256;
    int gridSize = (num + blockSize - 1) / blockSize;

    printf("Using blockSize = %d, gridSize = %d\n", blockSize, gridSize);

    printf("Starting timestamp -> random calculation...\n");

    multihash_kernel<<<gridSize, blockSize>>>(d_input, d_output, num);
    hipDeviceSynchronize();
    uint64_t end1 = get_time_in_nanosecond();

    uint64_t end = get_time_in_nanosecond();

    int limit = num - offset;

    printf("Total Time: %f ms\n", (end - start) / 1000000.0);
    // print speed per second
    printf("Speed: %f hashes per second\n", num / ((end1 - start) / 1000000000.0));

    printf("Limit: %d\n", limit);
    start = get_time_in_nanosecond();

    uint64_t start_enc = get_time_in_nanosecond();

    // set gridsize based on limit
    gridSize = (limit + blockSize - 1) / blockSize;

    for (size_t i = 0; i < offset_count; i++) 
    {
        printf("Starting encryption and search at offset %zu (%zu)\n", i, offsets[i]);
        encrypt_and_search_offset<<<gridSize, blockSize>>>(d_output, offsets[i], d_found, d_masks, d_matches, matches_size, limit);
        //check for error
        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
        {
            fprintf(stderr, "Error: %s\n", hipGetErrorString(err));
            return 1;
        }

        if (i > 0 && (i % 1) == 0)
        {
            uint64_t end_enc = get_time_in_nanosecond();
            printf("Enc Time: %f ms\n", (end_enc - start_enc) / 1000000.0);
            start_enc = get_time_in_nanosecond();
            printf("Progress: %zu/%zu (testing: %zu)\n", i, offset_count, offsets[i]);
            hipDeviceSynchronize();

            unsigned long long found = 0;
            hipMemcpy(&found, d_found, sizeof(unsigned long long), hipMemcpyDeviceToHost);
            if (found)
            {
                //unsigned long long encoded_offset_and_index = (unsigned long long)offset << 31 | idx << 1 | 1;

                size_t t_offset, t_index, match_pos;
                decode_offset_and_index(found, &t_offset, &t_index, &match_pos);

                printf("Found at  index %zu  offset=%zu ts = %zu + %zu filename = %s match_pos = %zu\n", i, t_offset, t_start, t_index, filename, match_pos);
                //write to output.txt
                FILE *f = fopen("output.txt", "a");
                if (f == NULL)
                {
                    printf("Error opening file!\n");
                    exit(1);
                }
                fprintf(f, "Found at  index %zu  offset=%zu ts = %zu + %zu filename = %s match_pos = %zu\n", i, t_offset, t_start, t_index, filename, match_pos);
                fclose(f);
                // reset for next
                hipMemcpy(d_found, &zero, sizeof(unsigned long long), hipMemcpyHostToDevice);
                // err = hipGetLastError();
                // if (err != hipSuccess)
                // {
                //     fprintf(stderr, "Error2: %s\n", hipGetErrorString(err));
                //     return 1;
                // }
            }
            //write checkpoint
            config["start_timestamp"] = t_start + i;
            config["index"] = i;
            std::ofstream o(checkpoint);
            o << std::setw(4) << config << std::endl;            
        }
    }

    hipDeviceSynchronize();

    long long found = 0;
    hipMemcpy(&found, d_found, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: %s\n", hipGetErrorString(err));
        return 1;
    }
    if (found)
    {
        size_t t_offset, t_index, match_pos;
        decode_offset_and_index(found, &t_offset, &t_index, &match_pos);
        printf("Found at  offset=%zu ts = %zu + %zu filename = %s match indxex = %zu\n",  t_offset, t_start, t_index, filename, match_pos);

    }

    uint64_t end_enc = get_time_in_nanosecond();
    printf("Enc Offset list Time: %f ms\n", (end_enc - start_enc) / 1000000.0);

    return 0;
}

int do_bruteforce_new(const char *filename)
{
    json config;

    std::string json_file = filename;

    std::string checkpoint = json_file + ".checkpoint.json";

    std::ifstream f(filename);
    if (!f.is_open())
    {
        fprintf(stderr, "Error: Could not open config file %s\n", filename);
        return 1;
    }

    try
    {
        f >> config;
    }
    catch (json::parse_error &e)
    {
        fprintf(stderr, "Error parsing JSON: %s\n", e.what());
        return 1;
    }

    size_t num;
    uint64_t t_start;
    size_t enc_count;
    size_t offset;

    size_t matches_size = 0;
    uint64_t *matches = 0;
    uint64_t *masks = 0;
    char **matches_filename = 0;

    try
    {
        t_start = config["start_timestamp"].get<uint64_t>(); // start T3

        num = config["count"].get<size_t>(); // stop at T3 + count ns

        offset = config["offset"].get<size_t>(); // we start at T3 + offset ns

        enc_count = config["brute_force_time_range"].get<size_t>(); // we stop at T3 + offset + enc_count ns

        // parse matches, and put it in array (match = plaintext^encrypted)
        if (config.contains("matches"))
        {
            matches_size = config["matches"].size();

            assert(matches_size < MAX_MATCHES);
            assert(matches_size > 0);

            matches = (uint64_t *)malloc(matches_size * sizeof(uint64_t));
            masks = (uint64_t *)malloc(matches_size * sizeof(uint64_t));
            matches_filename = (char **)malloc(matches_size * sizeof(char *));
            for (size_t i = 0; i < matches_size; i++)
            {
                uint64_t plaintext = std::stoull(config["matches"][i]["plaintext"].get<std::string>(), 0, 16);
                uint64_t encrypted = std::stoull(config["matches"][i]["encrypted"].get<std::string>(), 0, 16);
                uint64_t bitmask = std::stoull(config["matches"][i]["bitmask"].get<std::string>(), 0, 16);
                std::string filename = config["matches"][i]["filename"].get<std::string>();
                //strdup filename
                matches_filename[i] = strdup(filename.c_str());
                matches[i] = plaintext ^ encrypted;
                masks[i] = bitmask;
                printf("Match %zu: %016lx bitmask %016lx \n", i, matches[i], masks[i]);

            }
        }
    }
    catch (json::exception &e)
    {
        fprintf(stderr, "Error reading JSON values: %s\n", e.what());
        return 1;
    }

    printf("Configuration:\n");
    printf("num: %zu\n", num);
    printf("t_start: %lu\n", t_start);
    printf("t_end %lu\n", t_start + num);
    printf("enc_count: %zu\n", enc_count);
    printf("offset: %zu\n", offset);

    printf("Brute forcing: %zu enc count %zu\n", num, enc_count);

    // output
    size_t output_size = num * SHA256_DIGEST_SIZE * sizeof(uint8_t);
    printf("GPU RAM for random %.2f MB\n", output_size / 1024.0 / 1024.0);

    // Allocate device memory.
    uint8_t  *d_output;
    unsigned long long *d_found;
    unsigned long long *d_matches;
    unsigned long long *d_masks;

    if (hipMalloc(&d_output, output_size) != hipSuccess)
    {
        fprintf(stderr, "Error: hipMalloc failed for d_output\n");
        return 1;
    }
    if (hipMalloc(&d_found, sizeof(unsigned long long )) != hipSuccess)
    {
        fprintf(stderr, "Error: hipMalloc failed for d_found\n");
        return 1;
    }
    if (hipMalloc(&d_matches, matches_size * sizeof(uint64_t)) != hipSuccess)
    {
        fprintf(stderr, "Error: hipMalloc failed for d_matches\n");
        return 1;
    }
    if (hipMalloc(&d_masks, matches_size * sizeof(uint64_t)) != hipSuccess)
    {
        fprintf(stderr, "Error: hipMalloc failed for d_masks\n");
        return 1;
    }

    unsigned long long  zero = 0;
    hipMemcpy(d_found, &zero, sizeof(unsigned long long), hipMemcpyHostToDevice);

    // copy matches and masks
    hipMemcpy(d_matches, matches, matches_size * sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(d_masks, masks, matches_size * sizeof(uint64_t), hipMemcpyHostToDevice);

    uint64_t start = get_time_in_nanosecond();


    // Launch kernel.
    int blockSize = 256;
    int gridSize = (num + blockSize - 1) / blockSize;

    printf("Using blockSize = %d, gridSize = %d\n", blockSize, gridSize);

    printf("Starting timestamp -> random calculation...: %zu nanoseconds\n", num);

    multihash_kernel_in_memory<<<gridSize, blockSize>>>(d_output, t_start, num);

    hipDeviceSynchronize();
    uint64_t end1 = get_time_in_nanosecond();

    uint64_t end = get_time_in_nanosecond();


    int limit = num - enc_count - offset;

    printf("Total Time to translate timestamp to random %f ms\n", (end - start) / 1000000.0);
    // print speed per second
    printf("Speed: %f hashes per second\n", num / ((end1 - start) / 1000000000.0));

    printf("Limit: %d\n", limit);
    start = get_time_in_nanosecond();

    uint64_t start_enc = get_time_in_nanosecond();

    // set gridsize based on limit
    gridSize = (limit + blockSize - 1) / blockSize;

    uint64_t elapsed = 0;

    for (size_t i = 0; i < enc_count; i++)
    {
        // printf("Starting encryption and search at offset %d\n", i);
        encrypt_and_search_offset<<<gridSize, blockSize>>>(d_output, offset + i, d_found, d_masks, d_matches, matches_size, limit);
        // print error
        // hipError_t err = hipGetLastError();
        // if (err != hipSuccess)
        // {
        //     fprintf(stderr, "Error: %s\n", hipGetErrorString(err));
        //     return 1;
        // }
#define SKIP_CHECK 100
        if (i > 0 && (i % SKIP_CHECK) == 0)
        {
            uint64_t end_enc = get_time_in_nanosecond();
            printf("Enc Time (%zu): %f ms\n", i, (end_enc - start_enc) / 1000000.0);

            elapsed += (end_enc - start_enc);
            printf("%.2f  minutes elapsed, ", elapsed / 1000000000.0 / 60.0);
            //remaining
            double remaining = (enc_count - i) * ((end_enc - start_enc)/SKIP_CHECK);
            printf("%.2f  minutes remaining (%.2f hours). ", remaining / 1000000000.0 / 60.0, remaining / 1000000000.0 / 3600.0);

            start_enc = get_time_in_nanosecond();
            printf("Progress: %zu/%zu (testing: %zu)\n", i, enc_count, offset + i);
        
            hipDeviceSynchronize();
            // //print error
            // err = hipGetLastError();
            // if (err != hipSuccess)
            // {
            //     fprintf(stderr, "Error0: %s\n", hipGetErrorString(err));
            //     return 1;
            // }

            unsigned long long  found = 0;
            hipMemcpy(&found, d_found, sizeof(unsigned long long ), hipMemcpyDeviceToHost);
            // err = hipGetLastError();
            // if (err != hipSuccess)
            // {
            //     fprintf(stderr, "Error1: %s\n", hipGetErrorString(err));
            //     return 1;
            // }
            if (found)
            {
                size_t t_offset, t_index, match_pos;
                decode_offset_and_index(found, &t_offset, &t_index, &match_pos);
                printf("Found at  offset=%zu ts = %zu + %zu config_file = %s match_index %zu, file : %s\n",  t_offset, t_start, t_index, filename, match_pos, 
                    matches_filename[match_pos]);
        

                printf("Found at offset %zu found = %llu ts = %zu file = %s, match_index = %zu file: %s\n", t_offset, found, t_start + t_index, filename, match_pos, 
                    matches_filename[match_pos]);
                //write to output.txt
                FILE *f = fopen("output.txt", "a");
                if (f == NULL)
                {
                    printf("Error opening file!\n");
                    exit(1);
                }
                fprintf(f, "Found at offset %zu found = %llu ts = %zu file = %s, match_index = %zu file: %s\n", t_offset, found, t_start + t_index, filename, match_pos, 
                    matches_filename[match_pos]);
                fclose(f);
                // reset for next
                hipMemcpy(d_found, &zero, sizeof(unsigned long long ), hipMemcpyHostToDevice);
                // err = hipGetLastError();
                // if (err != hipSuccess)
                // {
                //     fprintf(stderr, "Error2: %s\n", hipGetErrorString(err));
                //     return 1;
                // }
            }
        
            //write checkpoint
            config["offset"] = offset + i;
            config["index"] = i;
            std::ofstream o(checkpoint);
            o << std::setw(4) << config << std::endl;            
        }
    }

    hipDeviceSynchronize();
    // //print error
    // err = hipGetLastError();
    // if (err != hipSuccess)
    // {
    //     fprintf(stderr, "Error0: %s\n", hipGetErrorString(err));
    //     return 1;
    // }

    unsigned long long found = 0;
    hipMemcpy(&found, d_found, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error1: %s\n", hipGetErrorString(err));
        return 1;
    }
    if (found)
    {
        size_t t_offset, t_index, match_pos;
        decode_offset_and_index(found, &t_offset, &t_index, &match_pos);
        printf("Found at  offset=%zu ts = %zu + %zu filename = %s\n",  t_offset, t_start, t_index, filename);

    }

    uint64_t end_enc = get_time_in_nanosecond();
    printf("Offset Enc Time: %f ms\n", (end_enc - start_enc) / 1000000.0);

    return 0;
}

int do_bruteforce(const char *filename)
{
    json config;

    std::string json_file = filename;

    std::string checkpoint = json_file + "checkpoint.json";

    std::ifstream f(filename);
    if (!f.is_open())
    {
        fprintf(stderr, "Error: Could not open config file %s\n", filename);
        return 1;
    }

    try
    {
        f >> config;
    }
    catch (json::parse_error &e)
    {
        fprintf(stderr, "Error parsing JSON: %s\n", e.what());
        return 1;
    }

    size_t num;
    uint64_t t_start;
    size_t enc_count;
    size_t offset;

    size_t matches_size = 0;
    uint64_t *matches = 0;
    uint64_t *masks = 0;

    try
    {
        t_start = config["start_timestamp"].get<uint64_t>(); // start T3

        num = config["count"].get<size_t>(); // stop at T3 + count ns

        offset = config["offset"].get<size_t>(); // we start at T3 + offset ns

        enc_count = config["brute_force_time_range"].get<size_t>(); // we stop at T3 + offset + enc_count ns

        // parse matches, and put it in array (match = plaintext^encrypted)
        if (config.contains("matches"))
        {
            matches_size = config["matches"].size();
            matches = (uint64_t *)malloc(matches_size * sizeof(uint64_t));
            masks = (uint64_t *)malloc(matches_size * sizeof(uint64_t));
            for (size_t i = 0; i < matches_size; i++)
            {
                uint64_t plaintext = std::stoull(config["matches"][i]["plaintext"].get<std::string>(), 0, 16);
                uint64_t encrypted = std::stoull(config["matches"][i]["encrypted"].get<std::string>(), 0, 16);
                uint64_t bitmask = std::stoull(config["matches"][i]["bitmask"].get<std::string>(), 0, 16);
                matches[i] = plaintext ^ encrypted;
                printf("Match %zu: %016lx\n", i, matches[i]);
                masks[i] = bitmask;
                printf("Mask %zu: %016lx\n", i, masks[i]);
            }
        }
    }
    catch (json::exception &e)
    {
        fprintf(stderr, "Error reading JSON values: %s\n", e.what());
        return 1;
    }

    printf("Configuration:\n");
    printf("num: %zu\n", num);
    printf("t_start: %lu\n", t_start);
    printf("enc_count: %zu\n", enc_count);
    printf("offset: %zu\n", offset);

    printf("Brute forcing: %zu enc count %zu\n", num, enc_count);
    size_t input_size = num * 19 * sizeof(uint8_t);
    printf("Input size: %.2f MB\n", input_size / 1024.0 / 1024.0);

    // Allocate host memory.
    uint8_t *h_input = (uint8_t *)malloc(input_size);

    // output
    size_t output_size = num * SHA256_DIGEST_SIZE * sizeof(uint8_t);
    printf("Output size: %.2f MB\n", output_size / 1024.0 / 1024.0);

    uint8_t *h_output = (uint8_t *)malloc(output_size);

    size_t enc_output_size = enc_count * TEST_ENCRYPT_BLOCK_SIZE * sizeof(uint8_t);

    uint8_t *h_output_enc = (uint8_t *)malloc(enc_output_size);

    uint64_t start_fill = get_time_in_nanosecond();

    fill_input(h_input, t_start, num); // GENERATE TS

    uint64_t end_fill = get_time_in_nanosecond();
    printf("Fill Time: %f ms\n", (end_fill - start_fill) / 1000000.0);

    // Allocate device memory.
    uint8_t *d_input, *d_output, *d_output_enc;
    int *d_found;
    unsigned long long *d_matches;
    unsigned long long *d_masks;

    if (hipMalloc(&d_input, input_size) != hipSuccess)
    {
        fprintf(stderr, "Error: hipMalloc failed for d_input\n");
        return 1;
    }
    if (hipMalloc(&d_output, output_size) != hipSuccess)
    {
        fprintf(stderr, "Error: hipMalloc failed for d_output\n");
        return 1;
    }
    if (hipMalloc(&d_output_enc, enc_output_size) != hipSuccess)
    {
        fprintf(stderr, "Error: hipMalloc failed for d_output_enc\n");
        return 1;
    }
    if (hipMalloc(&d_found, sizeof(int)) != hipSuccess)
    {
        fprintf(stderr, "Error: hipMalloc failed for d_found\n");
        return 1;
    }
    if (hipMalloc(&d_matches, matches_size * sizeof(uint64_t)) != hipSuccess)
    {
        fprintf(stderr, "Error: hipMalloc failed for d_matches\n");
        return 1;
    }
    if (hipMalloc(&d_masks, matches_size * sizeof(uint64_t)) != hipSuccess)
    {
        fprintf(stderr, "Error: hipMalloc failed for d_masks\n");
        return 1;
    }

    long zero = 0;
    hipMemcpy(d_found, &zero, sizeof(int), hipMemcpyHostToDevice);

    // copy matches and masks
    hipMemcpy(d_matches, matches, matches_size * sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(d_masks, masks, matches_size * sizeof(uint64_t), hipMemcpyHostToDevice);

    uint64_t start = get_time_in_nanosecond();

    hipMemcpy(d_input, h_input, input_size, hipMemcpyHostToDevice);

    // Launch kernel.
    int blockSize = 256;
    int gridSize = (num + blockSize - 1) / blockSize;

    printf("Using blockSize = %d, gridSize = %d\n", blockSize, gridSize);

    printf("Starting timestamp -> random calculation...\n");

    multihash_kernel<<<gridSize, blockSize>>>(d_input, d_output, num);
    hipDeviceSynchronize();
    uint64_t end1 = get_time_in_nanosecond();
    hipMemcpy(h_output, d_output, output_size, hipMemcpyDeviceToHost);

    uint64_t end = get_time_in_nanosecond();

    // //print first hash
    printf("First hash: ");
    for (int i = 0; i < 16; i++)
    {
        printf("%02x", h_output[i]);
    }
    printf("\n");
    // //print 2nd hash
    printf("+offset hash: ");
    for (int i = 0; i < 16; i++)
    {
        printf("%02x", h_output[SHA256_DIGEST_SIZE * offset + i]);
    }
    printf("\n");

    printf("Total Time: %f ms\n", (end - start) / 1000000.0);
    // print speed per second
    printf("Speed: %f hashes per second\n", num / ((end1 - start) / 1000000000.0));

    size_t limit = num - enc_count - offset;
    printf("Limit: %zu\n", limit);
    start = get_time_in_nanosecond();

    int gridSizeEnc = (enc_count + blockSize - 1) / blockSize;

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, gpuIndex);

    int maxThreadsPerBlock = prop.maxThreadsPerBlock;      // Maximum allowed per block
    blockSize = maxThreadsPerBlock;                        // Dynamically set blockSize
    gridSizeEnc = (enc_count + blockSize - 1) / blockSize; // Ensure full coverage

    printf("Encryption Using blockSize = %d, gridSize = %d\n", blockSize, gridSizeEnc);

    uint64_t start_enc = get_time_in_nanosecond();

    for (size_t i = 0; i < limit; i++)
    {

        // launch test_kcipher2_kernel_single
        uint32_t *host_i32 = (uint32_t *)(h_output + i * 32);

        uint32_t k1 = host_i32[0];
        uint32_t k2 = host_i32[1];
        uint32_t k3 = host_i32[2];
        uint32_t k4 = host_i32[3];


        if (i > 0 && i % 10000 == 0)
        {
            uint64_t end_enc = get_time_in_nanosecond();
            printf("10000 ns Time: %f ms\n", (end_enc - start_enc) / 1000000.0);
            printf("Speed for 10000 ns: %f enc per second\n", 10000 / ((end_enc - start_enc) / 1000000000.0));
            start_enc = get_time_in_nanosecond();

            printf("Processing %zu: T3 = %lu\n", i, t_start + i);
            fflush(stdout);
            // printf("Current key: %08x %08x %08x %08x\n", k1, k2, k3, k4);
        }

        if ((i % 100000) == 0)
        {
            // create checkpoint
            // modify json with current time
            config["start_timestamp"] = t_start + i;
            std::ofstream o(checkpoint);
            o << std::setw(4) << config << std::endl;
        }

        encrypt_and_search<<<gridSizeEnc, blockSize>>>(d_output + (offset + i) * 32, d_output_enc, k1, k2, k3, k4, d_found, d_masks, d_matches, matches_size, enc_count);

        hipDeviceSynchronize();

        // copy match
        int found;
        hipMemcpy(&found, d_found, sizeof(int), hipMemcpyDeviceToHost);
        if (found)
        {

            // only copy the large block if we found a match
            hipMemcpy(h_output_enc, d_output_enc, enc_output_size, hipMemcpyDeviceToHost);

            // reset found flag for next match
            hipMemcpy(d_found, &zero, sizeof(int), hipMemcpyHostToDevice);
            // reset d_output_enc
            hipMemset(d_output_enc, 0, enc_output_size);

            uint64_t *host_i64 = (uint64_t *)(h_output_enc);
            for (size_t j = 0; j < enc_count; j++)
            {
                for (size_t k = 0; k < matches_size; k++)
                {
                    if ((*host_i64 & masks[k]) == matches[k])
                    {
                        printf("Found Match %zu target %zu T3 = %lu T4 =  %lu offset=%lu\n", k, j, t_start + i, (t_start + i) + offset + j, offset + j);
                        // open file and write to it
                        FILE *out = fopen("output.txt", "a");
                        if (out != NULL)
                        {
			    fprintf(out, "Found Match %zu target %zu T3 = %lu T4 =  %lu offset=%lu\n", k, j, t_start + i, (t_start + i) + offset + j, offset + j);			
                            fclose(out);
                        }
                    }
                }

                host_i64 += 4;
            }
        }
    }

    end = get_time_in_nanosecond();

    printf("Total Time: %f ms\n", (end - start) / 1000000.0);
    // print speed per second
    printf("Speed: %f enc per second\n", enc_count / ((end - start) / 1000000000.0));

    // Cleanup.
    hipFree(d_input);
    hipFree(d_output);
    free(h_input);
    free(h_output);
    return 0;
}

//this will leak fd, but we don't care
uint8_t  *load_random(const char *filename, size_t *size)
{
    //mmap the file
    int fd;
    struct stat sb;
    uint8_t *addr;
    fd = open(filename, O_RDONLY);
    if (fd == -1)
    {
        perror("open");
        return 0;
    }
    //get file size using stat
    if (fstat(fd, &sb) == -1)
    {
        close(fd);
        perror("fstat");
        return 0;
    }
    *size = sb.st_size;
    addr = (uint8_t *)mmap(NULL, sb.st_size, PROT_READ, MAP_PRIVATE, fd, 0);
    if (addr == MAP_FAILED)
    {
        close(fd);
        perror("mmap");
        return 0;        
    }
    return addr;
}

uint8_t *hex_to_bytes(const char *search_hex)
{
    if (strlen(search_hex)!=32) {
        printf("Search hex must be 32 bytes hex characters\n");
        return 0;
    }

    uint8_t * search = (uint8_t *)malloc(16);

    //convert search_hex (32 hex character, no space) to search
    for (int i = 0; i < 16; i++)
    {
        char tmp[3];
        tmp[0] = search_hex[i * 2];
        tmp[1] = search_hex[i * 2 + 1];
        tmp[2] = 0;
        search[i] = strtol(tmp, NULL, 16);
    }

    printf("Searching: ");
    //hexdump to verify
    for (int i = 0; i < 16; i++)
    {
        printf("%02x", search[i]);
    }
    printf("\n");

    //for every 4 byte do: __builtin_bswap32
    uint32_t *search_i32 = (uint32_t *)search;
    for (int i = 0; i < 4; i++)
    {
        search_i32[i] = __builtin_bswap32(search_i32[i]);
    }
    return search;
}

int search_random(uint8_t  *addr, size_t size, const char *search_hex)
{
    if (!addr || !search_hex) {
        return -1;
    }

    uint8_t *search = hex_to_bytes(search_hex);
    if (!search) {
        return -1;
    }

    //now search the file using memmem
    void * pos = memmem(addr, size, search, 16);
    int found_pos = 1;
    if (pos ) {
        printf("Found at index %ld\n", ((uint8_t *)pos - addr)/16);
        found_pos = ((uint8_t *)pos - addr)/16;

    } else {
        printf("Not found\n");
    }
    return found_pos;
}

//hash computation in GPU, chacha8 in CPU
int  test_chacha8_speed(size_t num)
{
    printf("Test random + Chacha8 : %lu\n", num);
    size_t input_size = num * 19 * sizeof(uint8_t);
    printf("Input size: %.2f MB\n", input_size / 1024.0 / 1024.0);

    // Allocate host memory.
    uint8_t *h_input = (uint8_t *)malloc(input_size);

    uint64_t t_start = TEST_TIMESTAMP + 2000; // TEST:  0 is for chacha, +1000 for chacha_nonce

    // output
    size_t output_size = num * SHA256_DIGEST_SIZE * sizeof(uint8_t);
    printf("Output size: %.2f MB\n", output_size / 1024.0 / 1024.0);

    uint8_t *h_output = (uint8_t *)malloc(output_size);

    uint64_t start_fill = get_time_in_nanosecond();

    fill_input(h_input, t_start, num);
    uint64_t end_fill = get_time_in_nanosecond();
    printf("Fill Time: %f ms\n", (end_fill - start_fill) / 1000000.0);

    // Allocate device memory.
    uint8_t *d_input, *d_output;
    if (hipMalloc(&d_input, input_size) != hipSuccess)
    {
        fprintf(stderr, "Error: hipMalloc failed for d_input\n");
        return 1;
    }
    if (hipMalloc(&d_output, output_size) != hipSuccess)
    {
        fprintf(stderr, "Error: hipMalloc failed for d_output\n");
        return 1;
    }

    uint64_t start = get_time_in_nanosecond();

    hipMemcpy(d_input, h_input, input_size, hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (num + blockSize - 1) / blockSize; // Ensure full coverage

    printf("Using blockSize = %d, gridSize = %d\n", blockSize, gridSize);

    printf("Starting timestamp -> random calculation...\n");
    fflush(stdout);

    multihash_kernel<<<gridSize, blockSize>>>(d_input, d_output, num);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: %s\n", hipGetErrorString(err));
        return 1;
    }

    hipDeviceSynchronize();
    uint64_t end1 = get_time_in_nanosecond();
    hipMemcpy(h_output, d_output, output_size, hipMemcpyDeviceToHost);

    uint64_t end = get_time_in_nanosecond();
    printf("Total Time: %f ms\n", (end - start) / 1000000.0);
    // print speed per second
    printf("Speed: %f hashes per second\n", num / ((end1 - start) / 1000000000.0));
    // Cleanup.
    hipFree(d_input);
    hipFree(d_output);
    uint8_t input[64];
    uint8_t output[64];
    memset(input, 0, 64);
    start = get_time_in_nanosecond();
    for (size_t i = 0;  i < num; i++) {
        uint8_t *seed = h_output + i * SHA256_DIGEST_SIZE;
        uint8_t *iv = h_output + i * SHA256_DIGEST_SIZE;
        chacha8_ctx ctx;
        chacha8_keysetup(&ctx, seed, iv);
        chacha8_get_keystream_oneblock(&ctx,  output);
        
    }
    end = get_time_in_nanosecond();
    printf("Enc Total Time: %f ms\n", (end - start) / 1000000.0);
    //speed
    printf("Speed: %f enc per second\n", num / ((end - start) / 1000000000.0));


    return 0;
}

void hexdump(const char *title, const uint8_t *data, size_t size)
{
    printf("%s: ", title);
    const uint8_t *p = data;
    for (size_t i = 0; i < size; i++) {
        printf("%02x ", p[i]);
    }
    printf("\n");
}


int bruteforce_chacha(const char *filename)
{
    std::string json_file = filename;
    std::string checkpoint = json_file + "checkpoint.json";
    uint64_t t3_ts;
    uint64_t t3_t1_offset;
    uint64_t t1_t2_start_offset;
    uint64_t t1_t2_end_offset;
    uint64_t encrypted, plaintext, value;

    //parse JSON
    std::ifstream f(filename);
    if (!f.is_open())
    {
        fprintf(stderr, "Error: Could not open config file %s\n", filename);
        return 1;
    }
    json config;
    try
    {
        f >> config;
        t3_ts = config["t3_ts"].get<uint64_t>(); //start of kcipher2
        t3_t1_offset = config["t3_t1_offset"].get<uint64_t>(); //how far from T3 do we want to start our timestamp
        t1_t2_start_offset = config["t1_t2_start_offset"].get<uint64_t>(); //start offset to test for t1-t2
        t1_t2_end_offset = config["t1_t2_end_offset"].get<uint64_t>(); //end offset to test for t1-t2
        encrypted = std::stoull(config["encrypted"].get<std::string>(), 0, 16);
        printf("Encrypted: %016lx\n", encrypted);
        plaintext = std::stoull(config["plaintext"].get<std::string>(), 0, 16);
        printf("Plaintext: %016lx\n", plaintext);
        value = encrypted ^ plaintext;
        printf("Value: %016lx\n", value);
    }
    catch (json::parse_error &e)
    {
        fprintf(stderr, "Error parsing JSON: %s\n", e.what());
        return 1;
    }



    size_t num = t3_t1_offset + t1_t2_end_offset;

    size_t input_size = num * 19 * sizeof(uint8_t);
    printf("Input size: %.2f MB\n", input_size / 1024.0 / 1024.0);

    // Allocate host memory.
    uint8_t *h_input = (uint8_t *)malloc(input_size);

    uint64_t t_start = t3_ts - t3_t1_offset;
    printf("t_start: %lu\n", t_start);
    uint64_t t_end = t_start + num;
    printf("t_end: %lu\n", t_end);

    // output
    size_t output_size = num * SHA256_DIGEST_SIZE * sizeof(uint8_t);
    printf("Output size: %.2f MB\n", output_size / 1024.0 / 1024.0);

    uint8_t *h_output = (uint8_t *)malloc(output_size);

    uint64_t start_fill = get_time_in_nanosecond();

    fill_input(h_input, t_start, num);
    uint64_t end_fill = get_time_in_nanosecond();
    printf("Fill Time: %f ms\n", (end_fill - start_fill) / 1000000.0);

    // Allocate device memory.
    uint8_t *d_input, *d_output;
    if (hipMalloc(&d_input, input_size) != hipSuccess)
    {
        fprintf(stderr, "Error: hipMalloc failed for d_input\n");
        return 1;
    }
    if (hipMalloc(&d_output, output_size) != hipSuccess)
    {
        fprintf(stderr, "Error: hipMalloc failed for d_output\n");
        return 1;
    }

    uint64_t start = get_time_in_nanosecond();

    hipMemcpy(d_input, h_input, input_size, hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (num + blockSize - 1) / blockSize; // Ensure full coverage

    printf("Using blockSize = %d, gridSize = %d\n", blockSize, gridSize);

    printf("Starting seed calculation...\n");
    fflush(stdout);

    multihash_kernel_noswap<<<gridSize, blockSize>>>(d_input, d_output, num);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: %s\n", hipGetErrorString(err));
        return 1;
    }

    hipDeviceSynchronize();
    uint64_t end1 = get_time_in_nanosecond();
    hipMemcpy(h_output, d_output, output_size, hipMemcpyDeviceToHost);

    uint64_t end = get_time_in_nanosecond();
    printf("Total Time: %f ms\n", (end - start) / 1000000.0);
    // print speed per second
    printf("Speed: %f hashes per second\n", num / ((end1 - start) / 1000000000.0));
    // Cleanup.
    hipFree(d_input);

    size_t num_loop = t3_t1_offset;

    blockSize = 256;
    gridSize = (num_loop + blockSize - 1) / blockSize; // Ensure full coverage

    unsigned long long *d_found;
    if (hipMalloc(&d_found, sizeof(unsigned long long)) != hipSuccess)
    {
        fprintf(stderr, "Error: hipMalloc failed for d_found\n");
        return 1;
    }

    start = get_time_in_nanosecond();

    for (size_t offset = t1_t2_start_offset; offset < t1_t2_end_offset; offset++)
    {        
        size_t n = offset - t1_t2_start_offset;
        if ((n % 500)==0) {
            end1 = get_time_in_nanosecond();
            printf("Time: %f ms\n", (end1 - start) / 1000000.0);
            double offset_per_second = n / ((end1 - start) / 1000000000.0);
            printf("Speed: %f offsets per second\n", offset_per_second);
            printf("Processing offset %zu elapsed %f minutes\n", offset, (end1 - start) / 1000000000.0 / 60.0);
            //remaining
            double remaining = (t1_t2_end_offset - offset) / offset_per_second;
            printf("Remaining: %zu minutes\n", (size_t)(remaining / 60.0));
        }
#if 0
        printf("ts = %zu\n", t_start);
        printf("Tp Offset: %zu ts = %zu\n",  offset, t_start + offset);
        uint8_t output[64];
        uint64_t *val = (uint64_t *)output;
        for (size_t i = 0; i < num_loop; i++ ) { 
            uint8_t *seed = h_output + i * SHA256_DIGEST_SIZE;
            //hexdump("Seed", seed, 32);
            uint8_t *iv = h_output + (i + offset) * SHA256_DIGEST_SIZE;
            //hexdump("IV", iv, 16);
            chacha8_ctx ctx;
            chacha8_keysetup(&ctx, seed, iv);
            chacha8_get_keystream_oneblock(&ctx,  output);
            //hexdump output
            hexdump("Output", output, 64);
            if (*val == value) {
                printf("Found at time %zu offset = %zu\n", i, offset);
                //save to file output.txt
                FILE *out = fopen("output.txt", "a");
                if (out != NULL)
                {
                    fprintf(out, "Found at time %zu offset = %zu\n", i, offset);
                    fclose(out);
                }

                return 0;
            }
            //break;
        }
        //break;
#else 
        //test using GPU 
        
        chacha8_encrypt_and_match<<<gridSize, blockSize>>>(d_output, offset, d_found, value, num_loop);
        // //check for errors
        // hipError_t err = hipGetLastError();
        // if (err != hipSuccess)
        // {
        //     fprintf(stderr, "Error: %s\n", hipGetErrorString(err));
        //     return 1;
        // }
        //sync
        if (n % 1000 == 0) {
            hipDeviceSynchronize();
            unsigned long long found = 0;
            hipMemcpy(&found, d_found, sizeof(unsigned long long), hipMemcpyDeviceToHost);
            if (found) {
                //        unsigned long long encoded_offset_and_index = (unsigned long long)offset << 32 | idx;
                size_t t_offset, t_index;
                t_offset = found >> 32;
                t_index = found & 0xffffffff;
                printf("Found at offset %zu index %zu\n", t_offset, t_index + t_start);
                //write to file
                FILE *out = fopen("output.txt", "a");
                if (out != NULL)
                {
                    fprintf(out, "Found at offset %zu index %zu\n", t_offset, t_index + t_start);
                    fclose(out);
                }
                return 1;
            }
        }
#endif
    }

    return 0;

}



int main(int argc, char *argv[])
{
    size_t num = 128 * 1000;

    int smCount;
    hipDeviceGetAttribute(&smCount, hipDeviceAttributeMultiprocessorCount, gpuIndex);
    printf("SM count: %d\n", smCount);

    if (argc > 1)
    {
        if (strcmp(argv[1], "random") == 0) //test generate random from timestamp
        {
            if (argc > 2)
            {
                num = atoll(argv[2]);
            }

            return test_generate_random_only(num);
        }
        if (strcmp(argv[1], "random-gpu") == 0) //test generate random from timestamp, using GPU
        {
            if (argc > 2)
            {
                num = atoll(argv[2]);
            }

            return test_generate_random_only_in_gpu(num);
        }

        if (strcmp(argv[1], "saverandom") == 0)
        {
            uint64_t start_time = get_time_in_nanosecond();
            if (argc > 1)
            {
                start_time = atoll(argv[2]);
                char tmp[16];
                snprintf(tmp, sizeof(tmp), "%lu", start_time);
                if (strlen(tmp)<19) {
                    start_time = start_time * 1000000000;
                }

            }
            char filename[256];
            snprintf(filename, 256, "random_%lu.bin", start_time);

            return save_random(start_time, filename);
        }
        if (strcmp(argv[1], "search") == 0) {
            if (argc > 3) {
                const char *filename = argv[2];
                const char *search_hex = argv[3];
                size_t size;
                uint8_t *addr = load_random(filename, &size);
                if (addr) {
                    int res = search_random(addr, size, search_hex);
                    if (argc > 4) {
                        const char *search_hex2 = argv[4];
                        int res2 = search_random(addr, size, search_hex2);
                        printf("Diff: %d\n", res2-res);
                    }
                }
            } else {
                printf("usage: search random.bin 16_byte_hex_sequece_no_space\n");
            }
        }

        if (strcmp(argv[1], "enc") == 0) //test encryption only
        {
            if (argc > 2)
            {
                num = atoll(argv[2]);
            }
            return test_encryption_only(num);
        }

        if (strcmp(argv[1], "chacha8") == 0) //test chacha8 only
        {
            if (argc > 2)
            {
                num = atoll(argv[2]);
            }
            return test_chacha8_speed(num);
        }        

        if (strcmp(argv[1], "runchacha") == 0 || strcmp(argv[1], "runchacha8") == 0) //run chacha8 bruteforce
        {
            if (argc > 2)
            {                        
                if (argc > 3)
                {
                    gpuIndex = atoi(argv[3]);
                }

                int deviceCount;
                hipGetDeviceCount(&deviceCount);

                if (gpuIndex < 0 || gpuIndex >= deviceCount)
                {
                    printf("Invalid GPU index %d\n", gpuIndex);
                    return EXIT_FAILURE;
                }
                hipSetDevice(gpuIndex);
                printf("Using GPU %d\n", gpuIndex);

                return bruteforce_chacha(argv[2]);
            } else {
                printf("Usage: runchacha config.json\n");
            }
        }
        //main bruteforce loop
        if (strcmp(argv[1], "run") == 0 || strcmp(argv[1], "run2")==0  || strcmp(argv[1], "run3") == 0)
        {
            // read config from JSON file
            if (argc > 2)
            {
                if (argc > 3)
                {
                    gpuIndex = atoi(argv[3]);
                }

                int deviceCount;
                hipGetDeviceCount(&deviceCount);

                if (gpuIndex < 0 || gpuIndex >= deviceCount)
                {
                    printf("Invalid GPU index %d\n", gpuIndex);
                    return EXIT_FAILURE;
                }
                hipSetDevice(gpuIndex);
                printf("Using GPU %d\n", gpuIndex);

                if (strcmp(argv[1], "run") == 0)
                {
                    return do_bruteforce(argv[2]); //this is the slow method
                }
                else if (strcmp(argv[1], "run2") == 0)
                {
                    return do_bruteforce_new(argv[2]); //this is the faster method
                } else if (strcmp(argv[1], "run3") == 0) {
                    return do_bruteforce_offset(argv[2]); //this will read values from offset.txt instead of using ranges
                }
            }
            printf("Please specify the JSON config file\n");
        }
    }

    return 0;
}
